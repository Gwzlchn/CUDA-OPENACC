#include "hip/hip_runtime.h"
﻿#include "../include/Compute_On_GPU.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Init_First_Second.cuh"
#include "../include/Runge_Kutta.cuh"
#include "../include/Laser.cuh"

#include <cstdlib>
#include <hip/hip_runtime.h>


__global__ void pairs_init(particle_pair* pair_array, const size_t size,
                           const double min_r, const double min_p)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{

		distribution(pair_array[idx].first, pair_array[idx].second, idx, min_r, min_p);
	}
	return;
}


__global__ void pairs_first_step_on_gpu(particle_pair* first_setp_pair_array, const size_t size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx<size)
	{
		for (int i = 0; i < one_steps; i++)
			update_step_one(first_setp_pair_array[idx].first, first_setp_pair_array[idx].second);
	}


}

__global__ void pre_second_step_qq(double * QQ_array)
{
	size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		QQ_array[idx] = compute_qq_single(idx);
	}
}




__global__ void pre_second_step_E_arr_check
(const double* E1_array, const double* E2_array, double* E_check_array)
{
	size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E_check_array[idx] = compute_e_for_check(idx, E1_array[idx], E2_array[idx]);
	}
}



__global__ void pre_second_step_e1_arr(const double* QQ_array, const double EE0, double* E1_array)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E1_array[idx] = compute_e1_single(idx, QQ_array[idx], EE0);
	}

}


__global__ void pre_second_step_e2_arr(const double* QQ_array, const double EE0, double* E2_array)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E2_array[idx] = compute_e2_single(idx, QQ_array[idx], EE0);
	}

}



__global__ void pairs_second_step_on_gpu_every_step
(particle_pair* second_arr, const size_t size, double* E1_array, double* E2_array,
	particle_pair* every_step_arr)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;

	double4 e1_laser = make_double4(0.0, 0.0, 0.0, 0.0);
	double4 e2_laser = make_double4(0.0, 0.0, 0.0, 0.0);
	int idx_of_laser = -1; // 相当于nn
						   //double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4 = 0.0;
						   //double now_t = 0.0; //当前时间，相当于t(1)
	if (idx < size)
	{
		for (int i = 0; i < two_steps; i++)
		{

			if (idx_of_laser == -1)
			{
				e1_laser = make_double4(0.0, E1_array[0], E1_array[0], E1_array[1]);
				e2_laser = make_double4(0.0, E2_array[0], E2_array[0], E2_array[1]);
			}
			else
			{
				e1_laser = make_double4(E1_array[idx_of_laser], E1_array[idx_of_laser + 1], E1_array[idx_of_laser + 1], E1_array[idx_of_laser + 2]);
				e2_laser = make_double4(E2_array[idx_of_laser], E2_array[idx_of_laser + 1], E2_array[idx_of_laser + 1], E2_array[idx_of_laser + 2]);
			}
			idx_of_laser += 2;

			update_step_two(second_arr[0].first, second_arr[0].second,
				e1_laser, e2_laser);
			every_step_arr[i].first.x = second_arr[0].first.x;
			every_step_arr[i].first.y = second_arr[0].first.y;
			every_step_arr[i].first.z = second_arr[0].first.z;
			every_step_arr[i].first.px = second_arr[0].first.px;
			every_step_arr[i].first.py = second_arr[0].first.py;
			every_step_arr[i].first.pz = second_arr[0].first.pz;

			every_step_arr[i].second.x = second_arr[0].second.x;
			every_step_arr[i].second.y = second_arr[0].second.y;
			every_step_arr[i].second.z = second_arr[0].second.z;
			every_step_arr[i].second.px = second_arr[0].second.px;
			every_step_arr[i].second.py = second_arr[0].second.py;
			every_step_arr[i].second.pz = second_arr[0].second.pz;

		}

		
	}
}







__global__ void pairs_second_step_on_gpu
(particle_pair* second_arr, const size_t size, double* E1_array, double* E2_array)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	double4 e1_laser = make_double4(0.0, 0.0,0.0,0.0);
	double4 e2_laser = make_double4(0.0, 0.0,0.0,0.0);
	int idx_of_laser = -1; // 相当于nn
	//double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4 = 0.0;
	//double now_t = 0.0; //当前时间，相当于t(1)
	if (idx<size)
	{
		for (int i = 0; i < two_steps; i++)
		{
			
			if(idx_of_laser == -1 )
			{
				e1_laser = make_double4(0.0,E1_array[0],E1_array[0],E1_array[1]);
				e2_laser = make_double4(0.0,E2_array[0],E2_array[0],E2_array[1]);
			}
			else
			{
				e1_laser = make_double4(E1_array[idx_of_laser],E1_array[idx_of_laser + 1],E1_array[idx_of_laser + 1],E1_array[idx_of_laser +2]);
				e2_laser = make_double4(E2_array[idx_of_laser],E2_array[idx_of_laser + 1],E2_array[idx_of_laser + 1],E2_array[idx_of_laser +2]);
			}
			idx_of_laser += 2;
			
			update_step_two(second_arr[idx].first, second_arr[idx].second,
			                e1_laser, e2_laser);
			
			/*//第一个激光场强度
			t1 = now_t;
			if (t1 == 0)
			{
				e1_laser_t1 = 0.0;
				e2_laser_t1 = 0.0;
			}
			else
			{
				idx_of_ds = (2.0 * t1) / DX - 1;
				e1_laser_t1 = E1_array[idx_of_ds];
				e2_laser_t1 = E2_array[idx_of_ds];
			}
			//第二个激光场强度
			t2 = now_t + DX / 2.0;
			idx_of_ds = 2.0 * t2 / DX - 1;
			e1_laser_t2 = E1_array[idx_of_ds];
			e2_laser_t2 = E2_array[idx_of_ds];
			//第三个激光场强度
			t3 = now_t + DX / 2.0;
			idx_of_ds = 2 * t3 / DX - 1;
			e1_laser_t3 = E1_array[idx_of_ds];
			e2_laser_t3 = E2_array[idx_of_ds];
			//第四个激光场强度
			t4 = now_t + DX;
			idx_of_ds = 2.0 * t4 / DX - 1;
			e1_laser_t4 = E1_array[idx_of_ds];
			e2_laser_t4 = E2_array[idx_of_ds];
			double4 e1_laser = make_double4(e1_laser_t1, e1_laser_t2, e1_laser_t3, e1_laser_t4);
			double4 e2_laser = make_double4(e2_laser_t1, e2_laser_t2, e2_laser_t3, e2_laser_t4);
			
			now_t = now_t + DX;*/
			/**/

		}


	}
}



__global__ void pairs_second_step_on_gpu_fliter
(const particle_pair* second_step_pair_array, particle_pair* second_step_pair_array_filter,
 const size_t size, unsigned long long* count_z, unsigned long long* count_zz)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{
		double ee1 = CalculationE1(second_step_pair_array[idx].first, second_step_pair_array[idx].second);
		double ee2 = CalculationE2(second_step_pair_array[idx].first, second_step_pair_array[idx].second);



		if (ee1*ee2 < 0)
		{
			atomicAdd(count_z, 1);
		}
		if ((ee1 > 0) && (ee2 > 0))
		{
			size_t temp_idx = atomicAdd(count_zz, 1);
			/*nuclei temp;
			temp.first = second_arr[idx].first;
			temp.second = second_arr[idx].second;
			second__arr_filter[temp_idx - 1] = temp;*/
		}
	}

}
