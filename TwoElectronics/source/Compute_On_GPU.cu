#include "hip/hip_runtime.h"
﻿#include "../include/Compute_On_GPU.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Init_First_Second.cuh"
#include "../include/Runge_Kutta.cuh"
#include "../include/Laser.cuh"

#include <cstdlib>
#include <hip/hip_runtime.h>


__global__ void pairs_init(particle_pair* pair_array, const long size,
                           const double min_r, const double min_p)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{

		distribution(pair_array[idx].first, pair_array[idx].second, idx, min_r, min_p);
	}
	return;
}


__global__ void pairs_first_step_on_gpu(particle_pair* first_setp_pair_array, const long size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx<size)
	{
		for (int i = 0; i < one_steps; i++)
			update_step_one(first_setp_pair_array[idx].first, first_setp_pair_array[idx].second);
	}


}

__global__ void pre_second_step_qq_arr(double * QQ_array)
{
	unsigned long idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		QQ_array[idx] = compute_qq_single(idx);
	}
}




__global__ void pre_second_step_E_arr_check
(const double* E1_array, const double* E2_array, double* E_check_array)
{
	unsigned long idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E_check_array[idx] = compute_e_for_check(idx, E1_array[idx], E2_array[idx]);
	}
}



__global__ void pre_second_step_e1_arr(const double* QQ_array, const double EE0, double* E1_array)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E1_array[idx] = compute_e1_single(idx, QQ_array[idx], EE0);
	}

}


__global__ void pre_second_step_e2_arr(const double* QQ_array, const double EE0, double* E2_array)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E2_array[idx] = compute_e2_single(idx, QQ_array[idx], EE0);
	}

}




__global__ void pairs_second_step_on_gpu
(particle_pair* second_arr, const long size, double* E1_array, double* E2_array)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	double e1_laser_t1 = 0.0, e1_laser_t2 = 0.0, e1_laser_t3 = 0.0, e1_laser_t4 = 0.0;
	double e2_laser_t1 = 0.0, e2_laser_t2 = 0.0, e2_laser_t3 = 0.0, e2_laser_t4 = 0.0;
	int idx_of_ds = -1; // 相当于nn
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4 = 0.0;
	double now_t = 0.0; //当前时间，相当于t(1)
	if (idx<size)
	{
		for (int i = 0; i < two_steps; i++)
		{
			//第一个激光场强度
			t1 = now_t;
			if (t1 == 0)
			{
				e1_laser_t1 = 0.0;
				e2_laser_t1 = 0.0;
			}
			else
			{
				idx_of_ds = (2.0 * t1) / DX - 1;
				e1_laser_t1 = E1_array[idx_of_ds];
				e2_laser_t1 = E2_array[idx_of_ds];
			}
			//第二个激光场强度
			t2 = now_t + DX / 2.0;
			idx_of_ds = 2.0 * t2 / DX - 1;
			e1_laser_t2 = E1_array[idx_of_ds];
			e2_laser_t2 = E2_array[idx_of_ds];
			//第三个激光场强度
			t3 = now_t + DX / 2.0;
			idx_of_ds = 2 * t3 / DX - 1;
			e1_laser_t3 = E1_array[idx_of_ds];
			e2_laser_t3 = E2_array[idx_of_ds];
			//第四个激光场强度
			t4 = now_t + DX;
			idx_of_ds = 2.0 * t4 / DX - 1;
			e1_laser_t4 = E1_array[idx_of_ds];
			e2_laser_t4 = E2_array[idx_of_ds];
			double4 e1_laser = make_double4(e1_laser_t1, e1_laser_t2, e1_laser_t3, e1_laser_t4);
			double4 e2_laser = make_double4(e2_laser_t1, e2_laser_t2, e2_laser_t3, e2_laser_t4);
			update_step_two(second_arr[idx].first, second_arr[idx].second,
			                e1_laser, e2_laser);
			now_t = now_t + DX;
			/*if(idx_of_ds == -1 )
			update_step_two(second_arr[idx].first, second_arr[idx].second,
			0.0,DS[0],DS[0],DS[1]);
			else
			{
			update_step_two(second_arr[idx].first, second_arr[idx].second,
			DS[idx_of_ds], DS[idx_of_ds + 1], DS[idx_of_ds + 1], DS[idx_of_ds + 2]);
			}
			idx_of_ds += 2;*/

		}


	}
}



__global__ void pairs_second_step_on_gpu_fliter
(const particle_pair* second_step_pair_array, particle_pair* second_step_pair_array_filter,
 const long size, unsigned long long* count_z, unsigned long long* count_zz)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{
		double ee1 = CalculationE1(second_step_pair_array[idx].first, second_step_pair_array[idx].second);
		double ee2 = CalculationE2(second_step_pair_array[idx].first, second_step_pair_array[idx].second);

		if (ee1*ee2 < 0)
		{
			atomicAdd(count_z, 1);
		}
		if ((ee1 > 0) && (ee2 > 0))
		{
			unsigned long long temp_idx = atomicAdd(count_zz, 1);
			/*nuclei temp;
			temp.first = second_arr[idx].first;
			temp.second = second_arr[idx].second;
			second__arr_filter[temp_idx - 1] = temp;*/
		}
	}

}
