﻿#include "../include/Call_GPU.cuh"
#include "../include/Erorr_Check.hpp"
#include "../include/Init_First_Second.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Compute_On_GPU.cuh"
#include "../include/Laser.cuh"
#include <cstdio>
#include <hip/hip_runtime_api.h>

dim3 get_pre_block(int dimx)
{
	return dim3(dimx);
}

dim3 get_compute_block(int dimx)
{
	return dim3(dimx);
}

dim3 get_grid(long size, const dim3& block)
{
	return dim3((size + block.x - 1) / block.x, 1);
}





void Pairs_Init_Call_GPU(particle_pair * pair_array_gpu, const long size)
{
	//计算最小 r p;
	double min_r, min_p;
	get_min_r_min_p(NX_const, NY_const, min_r, min_p);


	
	dim3 block = get_pre_block();;
	dim3 grid = get_grid(size,block);
	pairs_init <<< grid, block >>> (pair_array_gpu, size, min_r, min_p);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Pairs_First_Steo_Call_GPU(particle_pair * pair_array_gpu, const long size)
{


	dim3 block = get_compute_block();;
	dim3 grid = get_grid(size, block);
	pairs_first_step_on_gpu <<< grid, block >>> (pair_array_gpu, size);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Prepare_Laser_QQ_array(double* qq_array_gpu)
{

	dim3 block = get_pre_block();;
	dim3 grid = get_grid((2 * two_steps), block);
	pre_second_step_qq << < grid, block >> > (qq_array_gpu);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}

void Pairs_Second_Step_Once_Call_GPU(particle_pair * pair_array_gpu,double* qq_array_gpu, const long size,const int index)
{
	double *gpu_e1, *gpu_e2;
	CHECK(hipMalloc((void **)(&gpu_e1), Bytes_Of_Array_Laser));
	CHECK(hipMalloc((void **)(&gpu_e2), Bytes_Of_Array_Laser));

	double EE0 = compute_ee0_by_index(index);
	dim3 pre_block = get_pre_block();
	
	dim3 pre_grid = get_grid((2*two_steps),pre_block);
	pre_second_step_e1_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e1);
	pre_second_step_e2_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e2);




}




