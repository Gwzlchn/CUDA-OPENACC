﻿#include "../include/Call_GPU.cuh"
#include "../include/Erorr_Check.hpp"
#include "../include/Init_First_Second.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Compute_On_GPU.cuh"
#include "../include/Laser.cuh"
#include "../include/PrintStruct.h"
#include <cstdio>
#include <hip/hip_runtime_api.h>

dim3 get_pre_block(int dimx)
{
	return dim3(dimx);
}

dim3 get_compute_block(int dimx)
{
	return dim3(dimx);
}

dim3 get_grid(size_t size, const dim3& block)
{
	return dim3((size + block.x - 1) / block.x, 1);
}

void SaveArraysWhichOnGPU(double* gpu_array, size_t size, const char* file_name)
{
	double* host_array = (double*)malloc(sizeof(double) * size);
	CHECK(hipMemcpy(host_array, gpu_array, sizeof(double) * size , hipMemcpyDeviceToHost));
	PrintArray(host_array, size, file_name);

}

void SavePairsWhichOnGPU(particle_pair* gpu_array, size_t size, const char* file_name)
{
	particle_pair* host_pairs = (particle_pair*)malloc(Bytes_Of_Pairs);
	CHECK(hipMemcpy(host_pairs, gpu_array, size, hipMemcpyDeviceToHost));
	PrintStruct(host_pairs, size, file_name);
}


void Pairs_Init_Call_GPU(particle_pair * pair_array_gpu, const size_t size)
{
	//计算最小 r p;
	double min_r, min_p;
	get_min_r_min_p(NX_const, NY_const, min_r, min_p);


	
	dim3 block = get_pre_block();;
	dim3 grid = get_grid(size,block);
	pairs_init <<< grid, block >>> (pair_array_gpu, size, min_r, min_p);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Pairs_First_Step_Call_GPU(particle_pair * pair_array_gpu, const size_t size)
{


	dim3 block = get_compute_block();;
	dim3 grid = get_grid(size, block);
	pairs_first_step_on_gpu <<< grid, block >>> (pair_array_gpu, size);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Prepare_Laser_QQ_array(double* qq_array_gpu)
{

	dim3 block = get_pre_block();;
	dim3 grid = get_grid((2 * two_steps), block);
	pre_second_step_qq << < grid, block >> > (qq_array_gpu);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}

void Pairs_Second_Step_Once_Call_GPU
(particle_pair * pair_array_first_step_gpu, double* qq_array_gpu, const size_t size, const int index, 
	size_t& count_z_once, size_t& count_zz_once)
{
	double *gpu_e1, *gpu_e2;
	CHECK(hipMalloc((void **)(&gpu_e1), Bytes_Of_Array_Laser));
	CHECK(hipMalloc((void **)(&gpu_e2), Bytes_Of_Array_Laser));

	double EE0 = compute_ee0_by_index(index);

	dim3 pre_block = get_pre_block();
	dim3 pre_grid = get_grid((2*two_steps),pre_block);
	pre_second_step_e1_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e1);
	pre_second_step_e2_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e2);

	//计算第二步循环
	particle_pair* second_array_gpu;
	dim3 com_block = get_compute_block();
	dim3 com_grid = get_grid(size, com_block);
	CHECK(hipMalloc((void **)(&second_array_gpu), Bytes_Of_Pairs));
	CHECK(hipMemcpy(second_array_gpu, pair_array_first_step_gpu, Bytes_Of_Pairs, hipMemcpyDeviceToDevice));
	pairs_second_step_on_gpu <<<com_grid,com_block>>> (second_array_gpu, size, gpu_e1, gpu_e2);


	//第二步循环后过滤
	particle_pair* second_array_filter_gpu;
	CHECK(hipMalloc((void **)(&second_array_filter_gpu), Bytes_Of_Pairs));
	size_t count_z, count_zz;
	Pairs_Second_Step_Filter_Call_GPU(second_array_gpu, second_array_filter_gpu, size, count_z, count_zz);
	count_z_once = count_z;
	count_zz_once = count_zz;



	CHECK(hipFree(second_array_gpu));
	CHECK(hipFree(second_array_filter_gpu));

	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	//释放内存


}

void Pairs_Second_Step_Filter_Call_GPU
(particle_pair * pair_array_sec_step_gpu, particle_pair * pair_array_filtered_gpu,
 size_t size, size_t& count_z, size_t& count_zz)
{
	count_z = 0;
	count_zz = 0;
	size_t *gpu_count_z_arr, *gpu_count_zz_arr;
	CHECK(hipMalloc((void**)&gpu_count_z_arr, size_ull));
	CHECK(hipMalloc((void**)&gpu_count_zz_arr, size_ull));

	dim3 com_block = get_compute_block();
	dim3 com_grid = get_grid(size, com_block);
	pairs_second_step_on_gpu_fliter << < com_grid, com_block, 0, 0 >> > (pair_array_sec_step_gpu,
		pair_array_filtered_gpu, size, gpu_count_z_arr , gpu_count_zz_arr );

	CHECK(hipMemcpy(&count_z , gpu_count_z_arr ,
		size_ull, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(&count_zz,gpu_count_zz_arr ,
		size_ull, hipMemcpyDeviceToHost));

}

void Pairs_Second_Step_Whole_Call_GPU(particle_pair* pair_array_gpu, const size_t size, const int iter_times)
{

	double* qq_array_gpu;
	CHECK(hipMalloc((void**)&qq_array_gpu, Bytes_Of_Array_Laser));
	Prepare_Laser_QQ_array(qq_array_gpu);

	//保存每次迭代的z,zz
	size_t* z_count_arr = new size_t[iter_times];
	size_t* zz_count_arr = new size_t[iter_times];
	//保存每次迭代的ee0
	double* ee0_arr = new double[iter_times];
	for(int i = 0;i<iter_times;i++)
	{
		ee0_arr[i]= compute_ee0_by_index(i);
		size_t z_once, zz_once;
		Pairs_Second_Step_Once_Call_GPU(pair_array_gpu, qq_array_gpu, size, i,
										z_once,zz_once);
		z_count_arr[i] = z_once;
		zz_count_arr[i] = zz_once;
	}
	Print_Count_Array(ee0_arr, z_count_arr, zz_count_arr, iter_times, ion_rate_file_name.c_str());

	CHECK(hipFree(qq_array_gpu));

	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());


}





