﻿#include "../include/Call_GPU.cuh"
#include "../include/Erorr_Check.hpp"
#include "../include/Init_First_Second.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Compute_On_GPU.cuh"
#include "../include/Laser.cuh"
#include "../include/PrintStruct.h"
#include <cstdio>
#include <hip/hip_runtime_api.h>

//typedef unsigned long long size_t;

dim3 get_pre_block(int dimx)
{
	return dim3(dimx);
}

dim3 get_compute_block(int dimx)
{
	return dim3(dimx);
}

dim3 get_grid(size_t size, const dim3& block)
{
	return dim3((size + block.x - 1) / block.x, 1);
}

void SaveArraysWhichOnGPU(double* gpu_array, size_t size, const char* file_name)
{
	double* host_array = (double*)malloc(sizeof(double) * size);
	CHECK(hipMemcpy(host_array, gpu_array, sizeof(double) * size , hipMemcpyDeviceToHost));
	PrintArray(host_array, size, file_name);

}

void SaveLaserArraysWhichOnGPU(double* e1_array, double* e2_array, double* e_check_array, size_t size,
	const char* file_name)
{
	size_t bytes_of_arr = sizeof(double) * size ;

	double* e1_host_array = (double*)malloc(bytes_of_arr);
	double* e2_host_array = (double*)malloc(bytes_of_arr);
	double* e_check_host_array = (double*)malloc(bytes_of_arr);

	CHECK(hipMemcpy(e1_host_array, e1_array, bytes_of_arr, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(e2_host_array, e2_array, bytes_of_arr, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(e_check_host_array, e_check_array, bytes_of_arr, hipMemcpyDeviceToHost));

	PrintLaserArrays(e1_host_array,e2_host_array,e_check_host_array, size, file_name);




}

void SavePairsWhichOnGPU(particle_pair* gpu_array, size_t size, const char* file_name)
{
	particle_pair* host_pairs = (particle_pair*)malloc(size * sizeof(particle_pair));
	CHECK(hipMemcpy(host_pairs, gpu_array, Bytes_Of_Pairs, hipMemcpyDeviceToHost));
	PrintStruct(host_pairs, size, file_name);
}


void Pairs_Init_Call_GPU(particle_pair * pair_array_gpu, const size_t size)
{
	//计算最小 r p;
	double min_r, min_p;
	get_min_r_min_p(NX_const, NY_const, min_r, min_p);


	
	dim3 block = get_pre_block();;
	dim3 grid = get_grid(size,block);
	pairs_init <<< grid, block >>> (pair_array_gpu, size, min_r, min_p);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Pairs_First_Step_Call_GPU(particle_pair * pair_array_gpu, const size_t size)
{


	dim3 block = get_compute_block();;
	dim3 grid = get_grid(size, block);
	pairs_first_step_on_gpu <<< grid, block >>> (pair_array_gpu, size);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Prepare_Laser_QQ_array(double* qq_array_gpu)
{

	dim3 block = get_pre_block();;
	dim3 grid = get_grid((2 * two_steps), block);
	pre_second_step_qq << < grid, block >> > (qq_array_gpu);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}

void Prepare_Laser_E1_array(double* qq_array_gpu,double* e1_array_gpu)
{
	dim3 block = get_pre_block();;
	dim3 grid = get_grid((2 * two_steps), block);

	pre_second_step_e1_arr << < grid, block >> > (qq_array_gpu,EE0_Check,e1_array_gpu);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void Prepare_Laser_E2_array(double* qq_array_gpu,double * e2_array_gpu)
{

	dim3 block = get_pre_block();;
	dim3 grid = get_grid((2 * two_steps), block);

	pre_second_step_e2_arr <<< grid, block >>> (qq_array_gpu, EE0_Check, e2_array_gpu);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}

void Prepare_Laser_E_Check_array(double* e1_array_gpu, double* e2_array_gpu, double* e_check_array_gpu)
{

	dim3 block = get_pre_block();;
	dim3 grid = get_grid((2 * two_steps), block);
	pre_second_step_E_arr_check << < grid, block >> > (e1_array_gpu,e2_array_gpu,e_check_array_gpu);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}

void Pairs_Second_Step_Once_Call_GPU
(particle_pair * pair_array_first_step_gpu, double* qq_array_gpu, const size_t size, const int index,
 unsigned long long& count_z_once, unsigned long long& count_zz_once)
{
	

	double EE0 = compute_ee0_by_index(index);
	Pairs_Second_Step_Once_Use_E0_Call_GPU(pair_array_first_step_gpu, qq_array_gpu, size, EE0,
		count_z_once, count_zz_once);




}








void Pairs_Second_Step_Filter_Call_GPU
(particle_pair * pair_array_sec_step_gpu, particle_pair * pair_array_filtered_gpu,
 size_t size, unsigned long long& count_z, unsigned long long& count_zz)
{
	count_z = 0;
	count_zz = 0;
	unsigned long long  *gpu_count_z, *gpu_count_zz;
	CHECK(hipMalloc((void**)&gpu_count_z, size_ull));
	CHECK(hipMalloc((void**)&gpu_count_zz, size_ull));

	dim3 com_block = get_compute_block();
	dim3 com_grid = get_grid(size, com_block);
	pairs_second_step_on_gpu_fliter << < com_grid, com_block, 0, 0 >> > (pair_array_sec_step_gpu,
	                                                                     pair_array_filtered_gpu, size, gpu_count_z , gpu_count_zz );

	
	
	
	CHECK(hipMemcpy(&count_z , gpu_count_z ,
		size_ull, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(&count_zz,gpu_count_zz,
		size_ull, hipMemcpyDeviceToHost));

}

void Pairs_Second_Step_Whole_Call_GPU(particle_pair* pair_array_gpu, const size_t size, const int iter_times)
{

	double* qq_array_gpu;
	CHECK(hipMalloc((void**)&qq_array_gpu, Bytes_Of_Array_Laser));
	Prepare_Laser_QQ_array(qq_array_gpu);

	//保存每次迭代的z,zz
	unsigned long long* z_count_arr = new unsigned long long[iter_times];
	unsigned long long* zz_count_arr = new unsigned long long[iter_times];
	//保存每次迭代的ee0
	double* ee0_arr = new double[iter_times];
	for(int i = 0;i<iter_times;i++)
	{
		ee0_arr[i]= compute_ee0_by_index(i);
		unsigned long long z_once, zz_once;
		Pairs_Second_Step_Once_Call_GPU(pair_array_gpu, qq_array_gpu, size, i,
		                                z_once,zz_once);
		z_count_arr[i] = z_once;
		zz_count_arr[i] = zz_once;
	}

	Print_Count_Array(ee0_arr, z_count_arr, zz_count_arr, iter_times, ion_rate_file_name.c_str());

	CHECK(hipFree(qq_array_gpu));

	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());


}



void Pairs_Second_Step_Once_Use_E0_Call_GPU(
	particle_pair * pair_array_first_step_gpu, double* qq_array_gpu, const size_t size, double EE0,
	unsigned long long& count_z_once, unsigned long long& count_zz_once)
{
	double *gpu_e1, *gpu_e2;
	CHECK(hipMalloc((void **)(&gpu_e1), Bytes_Of_Array_Laser));
	CHECK(hipMalloc((void **)(&gpu_e2), Bytes_Of_Array_Laser));

	//double EE0 = compute_ee0_by_index(index);
	//double EE0 = EE0_Check;
	dim3 pre_block = get_pre_block();
	dim3 pre_grid = get_grid((2 * two_steps), pre_block);
	pre_second_step_e1_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e1);
	pre_second_step_e2_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e2);

	//计算第二步循环
	particle_pair* second_array_gpu;
	dim3 com_block = get_compute_block();
	dim3 com_grid = get_grid(size, com_block);
	CHECK(hipMalloc((void **)(&second_array_gpu), Bytes_Of_Pairs));
	CHECK(hipMemcpy(second_array_gpu, pair_array_first_step_gpu, Bytes_Of_Pairs, hipMemcpyDeviceToDevice));
	pairs_second_step_on_gpu << <com_grid, com_block >> > (second_array_gpu, size, gpu_e1, gpu_e2);


	//第二步循环后过滤
	particle_pair* second_array_filter_gpu;
	CHECK(hipMalloc((void **)(&second_array_filter_gpu), Bytes_Of_Pairs));
	unsigned long long count_z, count_zz;
	Pairs_Second_Step_Filter_Call_GPU(second_array_gpu, second_array_filter_gpu, size, count_z, count_zz);
	count_z_once = count_z;
	count_zz_once = count_zz;

	SavePairsWhichOnGPU(second_array_filter_gpu, count_zz_once, second_step_file_name.c_str());

	//SavePairsWhichOnGPU(second_array_gpu,size,"OneStep.dat");

	CHECK(hipFree(second_array_gpu));
	CHECK(hipFree(second_array_filter_gpu));

	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());


}





void Pairs_Second_Step_Once(particle_pair* pair_array_gpu, const size_t size)
{

	double* qq_array_gpu;
	CHECK(hipMalloc((void**)&qq_array_gpu, Bytes_Of_Array_Laser));
	Prepare_Laser_QQ_array(qq_array_gpu);

	
		
	unsigned long long z_once, zz_once;
	Pairs_Second_Step_Once_Use_E0_Call_GPU(pair_array_gpu, qq_array_gpu, size, EE0_now,
		                                z_once,zz_once);
	
	


	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());


}



void every_step(int pairs)
{

	//申请显存空间
	particle_pair *pairs_array_single_step_gpu;
	CHECK(hipMalloc((void **)(&pairs_array_single_step_gpu), sizeof(particle_pair)));


	//计时
	double start = seconds();
	//计算
	Pairs_Init_Call_GPU(pairs_array_single_step_gpu, pairs);
	//保存
	SavePairsWhichOnGPU(pairs_array_single_step_gpu, pairs, init_file_name.c_str());
	//初始化完成
	double elapse = seconds();
	printf("Inition compltete %lf\n", elapse - start);




	double *gpu_e1, *gpu_e2, *qq_array_gpu;
	CHECK(hipMalloc((void **)(&gpu_e1), Bytes_Of_Array_Laser));
	CHECK(hipMalloc((void **)(&gpu_e2), Bytes_Of_Array_Laser));
	CHECK(hipMalloc((void **)(&qq_array_gpu), Bytes_Of_Array_Laser));

	//double EE0 = compute_ee0_by_index(index);
	double EE0 = EE0_Check;
	dim3 pre_block = get_pre_block();
	dim3 pre_grid = get_grid((2 * two_steps), pre_block);
	pre_second_step_qq << < pre_grid, pre_block >> > (qq_array_gpu);
	pre_second_step_e1_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e1);
	pre_second_step_e2_arr << < pre_grid, pre_block, 0, 0 >> > (qq_array_gpu, EE0, gpu_e2);

	particle_pair *pairs_array_every_step_gpu;
	CHECK(hipMalloc((void **)(&pairs_array_every_step_gpu), sizeof(particle_pair) * two_steps));
	start = seconds();
	//计算 后保存电离率
	//Pairs_Second_Step_Whole_Call_GPU(pairs_array_single_step_gpu, pairs, Iter_Count);
	dim3 block = get_compute_block();
	dim3 grid = get_grid(pairs, block);
	pairs_second_step_on_gpu_every_step << < 1,1 >> > (pairs_array_single_step_gpu, pairs, gpu_e1, gpu_e2, pairs_array_every_step_gpu);
	SavePairsWhichOnGPU(pairs_array_every_step_gpu, two_steps, "every_step.dat");
}
