#include "hip/hip_runtime.h"
﻿#include "../include/Init_First_Second.cuh"
#include "../include/Sci_Constant.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>





void get_min_r_min_p(int nx, int ny, double& min_r, double& min_p)
{
	double *R_Arr = (double*)malloc(nx * sizeof(double));
	double *P_Arr = (double*)malloc(ny * sizeof(double));

	for (int i = 0; i < nx; i++)
		R_Arr[i] = 0.5 + 0.01 * i;
	for (int i = 0; i < ny; i++)
		P_Arr[i] = 0.0 + 0.01*i;


	double** mat = new double*[nx];
	for (int i = 0; i<nx; i++)
		mat[i] = new double[ny];

	double Vh, Vk, Ek;
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			Vh = pow(Q_constant, 2) / (4.0*A_hardness*pow(R_Arr[i], 2)) *
				exp(A_hardness * (1.0 - pow((R_Arr[i] * P_Arr[j] / Q_constant), 4)));
			Vk = -2.0 / R_Arr[i];
			Ek = P_Arr[j] * P_Arr[j] / 2.0;
			mat[i][j] = Vh + Vk + Ek + 1.065;
		}
	}

	int min_x_index, min_y_index;
	double min = mat[0][0];
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			if (min > mat[i][j])
			{
				min = mat[i][j];
				min_x_index = i;
				min_y_index = j;
			}
		}
	}

	min_r = R_Arr[min_x_index];
	min_p = P_Arr[min_y_index];

	return;
}







__device__ void get_six_random(double2& two_random, double4& four_random, const int& seed)
{
	hiprandStatePhilox4_32_10_t s;
	hiprand_init(seed, 0, 0, &s);
	two_random = hiprand_uniform2_double(&s);
	four_random = make_double4(hiprand_uniform_double(&s), hiprand_uniform_double(&s),
	                           hiprand_uniform_double(&s), hiprand_uniform_double(&s));
	if (two_random.x == 1)
		two_random.x = 0;
	if (two_random.y == 1)
		two_random.y = 0;

	if (four_random.x == 1)
		four_random.x = 0;
	if (four_random.y == 1)
		four_random.y = 0;
	if (four_random.z == 1)
		four_random.z = 0;
	if (four_random.w == 1)
		four_random.w = 0;


}


__device__ void distribution(particle& first, particle& second,
                              int seed,  double min_r, double min_p)
{

	double2 two_rand;
	double4 four_rand;
	get_six_random(two_rand, four_rand, seed);

	double theta1 = two_rand.x * 2.0 * PI;
	double phi1 = two_rand.y * PI;


	first.x = min_r * sin(phi1) * cos(theta1);
	first.y = min_r * sin(phi1) * sin(theta1);
	first.z = min_r * cos(phi1);

	second.x = -first.x;
	second.y = -first.y;
	second.z = -first.z;


	double phi2 = four_rand.x * PI;
	double phi3 = four_rand.y * PI;
	double theta2 = four_rand.z * 2.0 * PI;
	double theta3 = four_rand.w * 2.0 * PI;

	first.px = min_p * cos(theta2)*sin(phi2);
	first.py = min_p * sin(theta2)*sin(phi2);
	first.pz = min_p * cos(phi2);

	second.px = min_p * cos(theta3)*sin(phi3);
	second.py = min_p * sin(theta3)*sin(phi3);
	second.pz = min_p * cos(phi3);
}


__device__  double nucleus_distance(const particle& first, const particle& second)
{
	return (pow((first.x - second.x), 2) + pow((first.y - second.y), 2) + pow((first.z - second.z), 2));
}


//第一个核，三个坐标的一阶导
__device__ double3 gx_gy_gz_first_nucleus(const particle& first, const particle& second)
{
	double Q_squre = pow(Q_constant, 2);
	//坐标平方和
	double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);

	//一阶导平方和
	double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);



	//第一个核 一阶导三个计算公式 对应 g1 g3 g5
	double3 gx_gy_gz;
	gx_gy_gz.x = first.px * (1.0 - 1.0 / Q_squre * loc_squre_sum_first * px_py_pz_squre_sum_first
		* exp(A_hardness * (1.0 - pow(loc_squre_sum_first * px_py_pz_squre_sum_first / Q_squre, 2))));

	gx_gy_gz.y = first.py * (1.0 - 1.0 / Q_squre * loc_squre_sum_first * px_py_pz_squre_sum_first
		* exp(A_hardness * (1.0 - pow(loc_squre_sum_first * px_py_pz_squre_sum_first / Q_squre, 2))));

	gx_gy_gz.z = first.pz * (1.0 - 1.0 / Q_squre * loc_squre_sum_first * px_py_pz_squre_sum_first
		* exp(A_hardness * (1.0 - pow(loc_squre_sum_first * px_py_pz_squre_sum_first / Q_squre, 2))));

	return gx_gy_gz;
}

//第二个核，三个坐标的一阶导
__device__ double3 gx_gy_gz_second_nucleus(const particle& first, const particle& second)
{
	const double Q_squre = pow(Q_constant, 2);

	//坐标平方和
	const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);

	//一阶导平方和
	const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);



	//第二个核 一阶导三个计算公式 对应 g2 g4 g6
	double3 gx_gy_gz;
	gx_gy_gz.x = second.px * (1.0 - 1.0 / Q_squre * loc_squre_sum_second * px_py_pz_squre_sum_second
		* exp(A_hardness * (1.0 - pow(loc_squre_sum_second * px_py_pz_squre_sum_second / Q_squre, 2))));

	gx_gy_gz.y = second.py * (1.0 - 1.0 / Q_squre * loc_squre_sum_second * px_py_pz_squre_sum_second
		* exp(A_hardness * (1.0 - pow(loc_squre_sum_second * px_py_pz_squre_sum_second / Q_squre, 2))));

	gx_gy_gz.z = second.pz * (1.0 - 1.0 / Q_squre * loc_squre_sum_second * px_py_pz_squre_sum_second
		* exp(A_hardness * (1.0 - pow(loc_squre_sum_second * px_py_pz_squre_sum_second / Q_squre, 2))));

	return gx_gy_gz;
}

//第一个核，三个坐标的二阶导
__device__ double3 fx_fy_fz_first_nucleus(const particle& first, const particle& second)
{
	const double Q_squre = pow(Q_constant, 2);

	//坐标平方和
	const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);
	//两核距离平方的1.5 次方,对应 sqrt(((z1-z2)**2.d0+(y1-y2)**2.d0+(x1-x2)**2.d0)**3.d0)
	const double distance_1_5_power = pow(distance_squre, 1.5);

	//坐标平方和的1.5 次方 ，对应 sqrt((z1**2.d0+y1**2.d0+x1**2.d0)**3.d0)
	const double loc_1_5_power_first = pow(loc_squre_sum_first, 1.5);
	const double loc_1_5_power_second = pow(loc_squre_sum_second, 1.5);

	//临时变量1 ,第一个粒子（一阶导平方和的平方 / Q方）
	const double temp1 = pow(px_py_pz_squre_sum_first, 2) / Q_squre;
	//临时变量2，第一个粒子（坐标平方和 * 一阶导平方和 / Q方）的平方
	const double temp2 = pow((loc_squre_sum_first * px_py_pz_squre_sum_first / Q_squre), 2);


	double3 fx_fy_fz;
	fx_fy_fz.x = first.x * ((Q_squre / 2.0 / A_hardness / pow(loc_squre_sum_first, 2) + temp1)
			* exp(A_hardness * (1.0 - temp2)) - 2.0 / loc_1_5_power_first)
		+ (first.x - second.x) / distance_1_5_power;

	fx_fy_fz.y = first.y * ((Q_squre / 2.0 / A_hardness / pow(loc_squre_sum_first, 2) + temp1)
			* exp(A_hardness * (1.0 - temp2)) - 2.0 / loc_1_5_power_first)
		+ (first.y - second.y) / distance_1_5_power;

	fx_fy_fz.z = first.z * ((Q_squre / 2.0 / A_hardness / pow(loc_squre_sum_first, 2) + temp1)
			* exp(A_hardness * (1.0 - temp2)) - 2.0 / loc_1_5_power_first)
		+ (first.z - second.z) / distance_1_5_power;

	return  fx_fy_fz;
}


//第二个核，三个坐标的二阶导
__device__ double3 fx_fy_fz_second_nucleus(const particle& first, const particle& second)
{
	const double Q_squre = pow(Q_constant, 2);

	//坐标平方和
	const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);
	//两核距离平方的1.5 次方,对应 sqrt(((z1-z2)**2.d0+(y1-y2)**2.d0+(x1-x2)**2.d0)**3.d0)
	const double distance_1_5_power = pow(distance_squre, 1.5);

	//坐标平方和的1.5 次方 ，对应 sqrt((z1**2.d0+y1**2.d0+x1**2.d0)**3.d0)
	const double loc_1_5_power_first = pow(loc_squre_sum_first, 1.5);
	const double loc_1_5_power_second = pow(loc_squre_sum_second, 1.5);

	//临时变量1 ,第二个粒子（一阶导平方和的平方 / Q方）
	//对应(pz1**2.d0+px1**2.d0+py1**2.d0)**2.d0/q**2.d0
	const double temp1 = pow(px_py_pz_squre_sum_second, 2) / Q_squre;
	//临时变量2，第二个粒子（坐标平方和 * 一阶导平方和 / Q方）的平方
	const double temp2 = pow((loc_squre_sum_second * px_py_pz_squre_sum_second / Q_squre), 2);


	double3 fx_fy_fz;
	fx_fy_fz.x = second.x * ((Q_squre / 2.0 / A_hardness / pow(loc_squre_sum_second, 2) + temp1)
			* exp(A_hardness * (1.0 - temp2)) - 2.0 / loc_1_5_power_second)
		- (first.x - second.x) / distance_1_5_power;

	fx_fy_fz.y = second.y * ((Q_squre / 2.0 / A_hardness / pow(loc_squre_sum_second, 2) + temp1)
			* exp(A_hardness * (1.0 - temp2)) - 2.0 / loc_1_5_power_second)
		- (first.y - second.y) / distance_1_5_power;

	fx_fy_fz.z = second.z * ((Q_squre / 2.0 / A_hardness / pow(loc_squre_sum_second, 2) + temp1)
			* exp(A_hardness * (1.0 - temp2)) - 2.0 / loc_1_5_power_second)
		- (first.z - second.z) / distance_1_5_power;

	return  fx_fy_fz;

}
