#include "hip/hip_runtime.h"
﻿#include "../include/Runge_Kutta.cuh"
#include "../include/Init_First_Second.cuh"
#include "../include/Sci_Constant.h"






//第一个粒子 K1~K4 第一步循环
__device__ derivative fisrt_k_one_to_four_fisrt_step(const particle& first, const particle& second)
{
	//二阶导 三个数
	const double3 first_fx = fx_fy_fz_first_nucleus(first, second);
	//一阶导 三个数
	const double3 first_gx = gx_gy_gz_first_nucleus(first, second);

	derivative first_px_fx;
	first_px_fx.px = first_gx.x;
	first_px_fx.py = first_gx.y;
	first_px_fx.pz = first_gx.z;

	first_px_fx.fx = first_fx.x;
	first_px_fx.fy = first_fx.y;
	first_px_fx.fz = first_fx.z;

	return first_px_fx;

}


__device__ derivative second_k_one_to_four_fisrt_step(const particle& first, const particle& second)
{
	//二阶导 三个数
	double3 second_fx = fx_fy_fz_second_nucleus(first, second);
	//一阶导 三个数
	double3 second_gx = gx_gy_gz_second_nucleus(first, second);
	derivative second_px_fx;
	second_px_fx.px = second_gx.x;
	second_px_fx.py = second_gx.y;
	second_px_fx.pz = second_gx.z;

	second_px_fx.fx = second_fx.x;
	second_px_fx.fy = second_fx.y;
	second_px_fx.fz = second_fx.z;
	return second_px_fx;

}



__device__ particle first_and_second_k_add_dx_raw(const derivative& k_one_to_four, const particle& raw_to_add)
{
	double now_dx = DX;

	particle k_add;
	k_add.x = raw_to_add.x + now_dx * k_one_to_four.px;
	k_add.y = raw_to_add.y + now_dx * k_one_to_four.py;
	k_add.z = raw_to_add.z + now_dx * k_one_to_four.pz;
	k_add.px = raw_to_add.px + now_dx * k_one_to_four.fx;
	k_add.py = raw_to_add.py + now_dx * k_one_to_four.fy;
	k_add.pz = raw_to_add.pz + now_dx * k_one_to_four.fz;

	return k_add;
}

__device__ particle first_and_second_k_add_dx_div(const derivative& k_one_to_four, const particle& raw_to_add)
{
	double now_dx = DX / 2.0;

	particle k_add;
	k_add.x = raw_to_add.x + now_dx * k_one_to_four.px;
	k_add.y = raw_to_add.y + now_dx * k_one_to_four.py;
	k_add.z = raw_to_add.z + now_dx * k_one_to_four.pz;
	k_add.px = raw_to_add.px + now_dx * k_one_to_four.fx;
	k_add.py = raw_to_add.py + now_dx * k_one_to_four.fy;
	k_add.pz = raw_to_add.pz + now_dx * k_one_to_four.fz;

	return k_add;
}






__device__ void k_one_to_four_add(const derivative& K1, const derivative& K2, const derivative& K3, const derivative& K4,
	particle& raw_to_add)
{
	raw_to_add.x = raw_to_add.x + DX * (K1.px + 2.0*K2.px + 2.0*K3.px + K4.px) / 6.0;
	raw_to_add.y = raw_to_add.y + DX * (K1.py + 2.0*K2.py + 2.0*K3.py + K4.py) / 6.0;
	raw_to_add.z = raw_to_add.z + DX * (K1.pz + 2.0*K2.pz + 2.0*K3.pz + K4.pz) / 6.0;
	raw_to_add.px = raw_to_add.px + DX * (K1.fx + 2.0*K2.fx + 2.0*K3.fx + K4.fx) / 6.0;
	raw_to_add.py = raw_to_add.py + DX * (K1.fy + 2.0*K2.fy + 2.0*K3.fy + K4.fy) / 6.0;
	raw_to_add.pz = raw_to_add.pz + DX * (K1.fz + 2.0*K2.fz + 2.0*K3.fz + K4.fz) / 6.0;



	return;
}





__device__ void update_step_one(particle& step_one_first, particle& step_one_second)
{
	//计算K1
	const derivative first_k1 = fisrt_k_one_to_four_fisrt_step(step_one_first, step_one_second);
	const derivative second_k1 = second_k_one_to_four_fisrt_step(step_one_first, step_one_second);
	const particle first_k1_add = first_and_second_k_add_dx_div(first_k1, step_one_first);
	const particle second_k1_add = first_and_second_k_add_dx_div(second_k1, step_one_second);

	//K2
	const derivative first_k2 = fisrt_k_one_to_four_fisrt_step(first_k1_add, second_k1_add);
	const derivative second_k2 = second_k_one_to_four_fisrt_step(first_k1_add, second_k1_add);
	const particle first_k2_add = first_and_second_k_add_dx_div(first_k2, step_one_first);
	const particle second_k2_add = first_and_second_k_add_dx_div(second_k2, step_one_second);

	//K3
	const derivative first_k3 = fisrt_k_one_to_four_fisrt_step(first_k2_add, second_k2_add);
	const derivative second_k3 = second_k_one_to_four_fisrt_step(first_k2_add, second_k2_add);
	const particle first_k3_add = first_and_second_k_add_dx_raw(first_k3, step_one_first);
	const particle second_k3_add = first_and_second_k_add_dx_raw(second_k3, step_one_second);

	//K4
	const derivative first_k4 = fisrt_k_one_to_four_fisrt_step(first_k3_add, second_k3_add);
	const derivative second_k4 = second_k_one_to_four_fisrt_step(first_k3_add, second_k3_add);

	k_one_to_four_add(first_k1, first_k2, first_k3, first_k4, step_one_first);
	k_one_to_four_add(second_k1, second_k2, second_k3, second_k4, step_one_second);

	return;
}





//第一个粒子 K1~K4 第二步循环
__device__ derivative fisrt_k_one_to_four_second_step
(const particle& first, const particle& second, const double& e1_laser, const double& e2_laser)
{
	const double3 first_fx = fx_fy_fz_first_nucleus(first, second);
	const double3 first_gx = gx_gy_gz_first_nucleus(first, second);
	derivative first_px_fx;
	first_px_fx.px = first_gx.x;
	first_px_fx.py = first_gx.y;
	first_px_fx.pz = first_gx.z;
	first_px_fx.fx = first_fx.x;
	first_px_fx.fy = first_fx.y - e2_laser;
	first_px_fx.fz = first_fx.z - e1_laser;

	return first_px_fx;

}

//第二个粒子 K1~K4 第二步循环
__device__ derivative second_k_one_to_four_second_step
(const particle& first, const particle& second, const double& e1_laser, const double& e2_laser)
{
	const double3 second_fx = fx_fy_fz_second_nucleus(first, second);
	const double3 second_gx = gx_gy_gz_second_nucleus(first, second);
	derivative second_px_fx;
	second_px_fx.px = second_gx.x;
	second_px_fx.py = second_gx.y;
	second_px_fx.pz = second_gx.z;
	second_px_fx.fx = second_fx.x;
	second_px_fx.fy = second_fx.y - e2_laser;
	second_px_fx.fz = second_fx.z - e1_laser;
	return second_px_fx;

}







__device__ void update_step_two(particle& step_one_first, particle& step_one_second,
	const double4 e1_laser_now, const double4 e2_laser_now)
{
	//计算K1
	const derivative first_k1 = fisrt_k_one_to_four_second_step(step_one_first, step_one_second, e1_laser_now.x, e2_laser_now.x);
	const derivative second_k1 = second_k_one_to_four_second_step(step_one_first, step_one_second, e1_laser_now.x, e2_laser_now.x);
	const particle first_k1_add = first_and_second_k_add_dx_div(first_k1, step_one_first);
	const particle second_k1_add = first_and_second_k_add_dx_div(second_k1, step_one_second);

	//K2
	const derivative first_k2 = fisrt_k_one_to_four_second_step(first_k1_add, second_k1_add, e1_laser_now.y, e2_laser_now.y);
	const derivative second_k2 = second_k_one_to_four_second_step(first_k1_add, second_k1_add, e1_laser_now.y, e2_laser_now.y);
	const particle first_k2_add = first_and_second_k_add_dx_div(first_k2, step_one_first);
	const particle second_k2_add = first_and_second_k_add_dx_div(second_k2, step_one_second);

	//K3
	const derivative first_k3 = fisrt_k_one_to_four_second_step(first_k2_add, second_k2_add, e1_laser_now.z, e2_laser_now.z);
	const derivative second_k3 = second_k_one_to_four_second_step(first_k2_add, second_k2_add, e1_laser_now.z, e2_laser_now.z);
	const particle first_k3_add = first_and_second_k_add_dx_raw(first_k3, step_one_first);
	const particle second_k3_add = first_and_second_k_add_dx_raw(second_k3, step_one_second);

	//K4
	const derivative first_k4 = fisrt_k_one_to_four_second_step(first_k3_add, second_k3_add, e1_laser_now.w, e2_laser_now.w);
	const derivative second_k4 = second_k_one_to_four_second_step(first_k3_add, second_k3_add, e1_laser_now.w, e2_laser_now.w);

	k_one_to_four_add(first_k1, first_k2, first_k3, first_k4, step_one_first);
	k_one_to_four_add(second_k1, second_k2, second_k3, second_k4, step_one_second);
}

