#include "hip/hip_runtime.h"
﻿#include "../include/Runge_Kutta.cuh"
#include "../include/Init_First_Second.cuh"
#include "../include/Sci_Constant.h"






//第一个粒子 K1~K4 第一步循环
__device__ derivative first_k_one_to_four_first_step(const particle& first, const particle& second , const double& t)
{
	//二阶导 三个数
	const double3 first_fx = fx_fy_fz_first_nucleus(first, second,t);
	//一阶导 三个数
	const double3 first_gx = gx_gy_gz_first_nucleus(first, second,t);

	derivative first_px_fx;
	first_px_fx.px = first_gx.x;
	first_px_fx.py = first_gx.y;
	first_px_fx.pz = first_gx.z;

	first_px_fx.fx = first_fx.x;
	first_px_fx.fy = first_fx.y;
	first_px_fx.fz = first_fx.z;
	
	return first_px_fx;

}


__device__ derivative second_k_one_to_four_first_step(const particle& first, const particle& second, const double& t)
{
	//二阶导 三个数
	double3 second_fx = fx_fy_fz_second_nucleus(first, second,t);
	//一阶导 三个数
	double3 second_gx = gx_gy_gz_second_nucleus(first, second,t);
	derivative second_px_fx;
	second_px_fx.px = second_gx.x;
	second_px_fx.py = second_gx.y;
	second_px_fx.pz = second_gx.z;

	second_px_fx.fx = second_fx.x;
	second_px_fx.fy = second_fx.y;
	second_px_fx.fz = second_fx.z;
	return second_px_fx;

}



__device__ particle first_and_second_k_add_dx_raw(const derivative& k_one_to_four, const particle& raw_to_add)
{
	double now_dx = DX;

	particle k_add;
	k_add.x = raw_to_add.x + now_dx * k_one_to_four.px;
	k_add.y = raw_to_add.y + now_dx * k_one_to_four.py;
	k_add.z = raw_to_add.z + now_dx * k_one_to_four.pz;
	k_add.px = raw_to_add.px + now_dx * k_one_to_four.fx;
	k_add.py = raw_to_add.py + now_dx * k_one_to_four.fy;
	k_add.pz = raw_to_add.pz + now_dx * k_one_to_four.fz;

	return k_add;
}

__device__ particle first_and_second_k_add_dx_div(const derivative& k_one_to_four, const particle& raw_to_add)
{
	double now_dx = DX / 2.0;

	particle k_add;
	k_add.x = raw_to_add.x + now_dx * k_one_to_four.px;
	k_add.y = raw_to_add.y + now_dx * k_one_to_four.py;
	k_add.z = raw_to_add.z + now_dx * k_one_to_four.pz;
	k_add.px = raw_to_add.px + now_dx * k_one_to_four.fx;
	k_add.py = raw_to_add.py + now_dx * k_one_to_four.fy;
	k_add.pz = raw_to_add.pz + now_dx * k_one_to_four.fz;

	return k_add;
}






__device__ void k_one_to_four_add(const derivative& K1, const derivative& K2, const derivative& K3, const derivative& K4,
	particle& raw_to_add)
{
	raw_to_add.x = raw_to_add.x + DX * (K1.px + 2.0*K2.px + 2.0*K3.px + K4.px) / 6.0;
	raw_to_add.y = raw_to_add.y + DX * (K1.py + 2.0*K2.py + 2.0*K3.py + K4.py) / 6.0;
	raw_to_add.z = raw_to_add.z + DX * (K1.pz + 2.0*K2.pz + 2.0*K3.pz + K4.pz) / 6.0;
	raw_to_add.px = raw_to_add.px + DX * (K1.fx + 2.0*K2.fx + 2.0*K3.fx + K4.fx) / 6.0;
	raw_to_add.py = raw_to_add.py + DX * (K1.fy + 2.0*K2.fy + 2.0*K3.fy + K4.fy) / 6.0;
	raw_to_add.pz = raw_to_add.pz + DX * (K1.fz + 2.0*K2.fz + 2.0*K3.fz + K4.fz) / 6.0;



	return;
}


__device__ void fill_every_step(particle& init_step_first,particle& init_step_second,particle_pair& every_step_arr,const double& t)
{
	double time = t;

	//计算K1
	const derivative first_k1 = first_k_one_to_four_first_step(init_step_first, init_step_second, time);
	const derivative second_k1 = second_k_one_to_four_first_step(init_step_first, init_step_second,time);
	const particle first_k1_add = first_and_second_k_add_dx_div(first_k1, init_step_first);
	const particle second_k1_add = first_and_second_k_add_dx_div(second_k1, init_step_second);
	time = t + DX / 2.0;
	//K2
	const derivative first_k2 = first_k_one_to_four_first_step(first_k1_add, second_k1_add, time);
	const derivative second_k2 = second_k_one_to_four_first_step(first_k1_add, second_k1_add, time);
	const particle first_k2_add = first_and_second_k_add_dx_div(first_k2, init_step_first);
	const particle second_k2_add = first_and_second_k_add_dx_div(second_k2, init_step_second);

	//K3
	const derivative first_k3 = first_k_one_to_four_first_step(first_k2_add, second_k2_add, time);
	const derivative second_k3 = second_k_one_to_four_first_step(first_k2_add, second_k2_add, time);
	const particle first_k3_add = first_and_second_k_add_dx_raw(first_k3, init_step_first);
	const particle second_k3_add = first_and_second_k_add_dx_raw(second_k3, init_step_second);
	time = t + DX;
	//K4
	const derivative first_k4 = first_k_one_to_four_first_step(first_k3_add, second_k3_add, time);
	const derivative second_k4 = second_k_one_to_four_first_step(first_k3_add, second_k3_add, time);

	k_one_to_four_add(first_k1, first_k2, first_k3, first_k4, init_step_first);
	k_one_to_four_add(second_k1, second_k2, second_k3, second_k4, init_step_second);


	every_step_arr.first.x = init_step_first.x;
	every_step_arr.first.y = init_step_first.y;
	every_step_arr.first.z = init_step_first.z;
	every_step_arr.first.px = init_step_first.px;
	every_step_arr.first.py = init_step_first.py;
	every_step_arr.first.pz = init_step_first.pz;


	every_step_arr.second.x = init_step_second.x;
	every_step_arr.second.y = init_step_second.y;
	every_step_arr.second.z = init_step_second.z;
	every_step_arr.second.px = init_step_second.px;
	every_step_arr.second.py = init_step_second.py;
	every_step_arr.second.pz = init_step_second.pz;

	return;
}


__device__ void update_step_one(particle& step_one_first, particle& step_one_second, const double& t)
{
	double time = t;
	//计算K1
	const derivative first_k1 = first_k_one_to_four_first_step(step_one_first, step_one_second, time);
	const derivative second_k1 = second_k_one_to_four_first_step(step_one_first, step_one_second, time);
	const particle first_k1_add = first_and_second_k_add_dx_div(first_k1, step_one_first);
	const particle second_k1_add = first_and_second_k_add_dx_div(second_k1, step_one_second);
	time = t + DX / 2.0;
	//K2
	const derivative first_k2 = first_k_one_to_four_first_step(first_k1_add, second_k1_add, time);
	const derivative second_k2 = second_k_one_to_four_first_step(first_k1_add, second_k1_add, time);
	const particle first_k2_add = first_and_second_k_add_dx_div(first_k2, step_one_first);
	const particle second_k2_add = first_and_second_k_add_dx_div(second_k2, step_one_second);

	//K3
	const derivative first_k3 = first_k_one_to_four_first_step(first_k2_add, second_k2_add, time);
	const derivative second_k3 = second_k_one_to_four_first_step(first_k2_add, second_k2_add, time);
	const particle first_k3_add = first_and_second_k_add_dx_raw(first_k3, step_one_first);
	const particle second_k3_add = first_and_second_k_add_dx_raw(second_k3, step_one_second);
	time = t + DX;
	//K4
	const derivative first_k4 = first_k_one_to_four_first_step(first_k3_add, second_k3_add, time);
	const derivative second_k4 = second_k_one_to_four_first_step(first_k3_add, second_k3_add, time);

	k_one_to_four_add(first_k1, first_k2, first_k3, first_k4, step_one_first);
	k_one_to_four_add(second_k1, second_k2, second_k3, second_k4, step_one_second);

	return;
}





//第一个粒子 K1~K4 第二步循环
__device__ derivative first_k_one_to_four_second_step
(const particle& first, const particle& second, const double& e1_laser, const double& e2_laser, const double& t)
{
	const double3 first_fx = fx_fy_fz_first_nucleus(first, second,t);
	const double3 first_gx = gx_gy_gz_first_nucleus(first, second, t);
	derivative first_px_fx;
	first_px_fx.px = first_gx.x;
	first_px_fx.py = first_gx.y;
	first_px_fx.pz = first_gx.z;
	first_px_fx.fx = first_fx.x;
	first_px_fx.fy = first_fx.y - e2_laser;
	first_px_fx.fz = first_fx.z - e1_laser;

	return first_px_fx;

}

//第二个粒子 K1~K4 第二步循环
__device__ derivative second_k_one_to_four_second_step
(const particle& first, const particle& second, const double& e1_laser, const double& e2_laser, const double& t)
{
	const double3 second_fx = fx_fy_fz_second_nucleus(first, second, t);
	const double3 second_gx = gx_gy_gz_second_nucleus(first, second, t);
	derivative second_px_fx;
	second_px_fx.px = second_gx.x;
	second_px_fx.py = second_gx.y;
	second_px_fx.pz = second_gx.z;
	second_px_fx.fx = second_fx.x;
	second_px_fx.fy = second_fx.y - e2_laser;
	second_px_fx.fz = second_fx.z - e1_laser;

	return second_px_fx;

}







__device__ void update_step_two(particle& step_one_first, particle& step_one_second,
	const double4 e1_laser_now, const double4 e2_laser_now, const double& t)
{
	double time = t;
	//计算K1
	const derivative first_k1 = first_k_one_to_four_second_step(step_one_first, step_one_second, e1_laser_now.x, e2_laser_now.x,time);
	const derivative second_k1 = second_k_one_to_four_second_step(step_one_first, step_one_second, e1_laser_now.x, e2_laser_now.x, time);
	const particle first_k1_add = first_and_second_k_add_dx_div(first_k1, step_one_first);
	const particle second_k1_add = first_and_second_k_add_dx_div(second_k1, step_one_second);
	time = t + DX / 2.0;
	//K2
	const derivative first_k2 = first_k_one_to_four_second_step(first_k1_add, second_k1_add, e1_laser_now.y, e2_laser_now.y, time);
	const derivative second_k2 = second_k_one_to_four_second_step(first_k1_add, second_k1_add, e1_laser_now.y, e2_laser_now.y, time);
	const particle first_k2_add = first_and_second_k_add_dx_div(first_k2, step_one_first);
	const particle second_k2_add = first_and_second_k_add_dx_div(second_k2, step_one_second);

	//K3
	const derivative first_k3 = first_k_one_to_four_second_step(first_k2_add, second_k2_add, e1_laser_now.z, e2_laser_now.z, time);
	const derivative second_k3 = second_k_one_to_four_second_step(first_k2_add, second_k2_add, e1_laser_now.z, e2_laser_now.z, time);
	const particle first_k3_add = first_and_second_k_add_dx_raw(first_k3, step_one_first);
	const particle second_k3_add = first_and_second_k_add_dx_raw(second_k3, step_one_second);
	time = t + DX;
	//K4
	const derivative first_k4 = first_k_one_to_four_second_step(first_k3_add, second_k3_add, e1_laser_now.w, e2_laser_now.w, time);
	const derivative second_k4 = second_k_one_to_four_second_step(first_k3_add, second_k3_add, e1_laser_now.w, e2_laser_now.w, time);

	k_one_to_four_add(first_k1, first_k2, first_k3, first_k4, step_one_first);
	k_one_to_four_add(second_k1, second_k2, second_k3, second_k4, step_one_second);
}

