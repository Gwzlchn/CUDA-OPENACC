#include "hip/hip_runtime.h"
﻿
#include "../include/Laser.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Runge_Kutta.cuh"

#include <cmath>









__device__ double CalculationE1(const particle& first, const particle& second)
{
	//坐标平方和
	const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	//const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	//const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);


	return  0.5 * px_py_pz_squre_sum_first - 2.0 / sqrt(loc_squre_sum_first) +
		1.0 / loc_squre_sum_first * Q_constant * Q_constant / 4.0 / A_hardness *
		exp(A_hardness * (1.0 - pow((loc_squre_sum_first * px_py_pz_squre_sum_first /
			                            Q_constant * Q_constant), 2))) +
		1.0 / sqrt(distance_squre) / 2.0;
}

__device__ double CalculationE2(const particle& first, const particle& second)
{
	//坐标平方和
	//const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	//const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);


	return  0.5 * px_py_pz_squre_sum_second - 2.0 / sqrt(loc_squre_sum_second) +
		1.0 / loc_squre_sum_second * Q_constant * Q_constant / 4.0 / A_hardness *
		exp(A_hardness * (1.0 - pow((loc_squre_sum_second * px_py_pz_squre_sum_second /
			                            Q_constant * Q_constant), 2))) +
		1.0 / sqrt(distance_squre) / 2.0;
}









__device__ double compute_qq_single(const size_t& now_step)
{
	double t1 = 0.5 * DX * (now_step + 1);
	return  pow((sin(Omega1 / 2.0 / (2 * N1_const + N2_const)*t1)), 2);

}

__device__ double compute_e_for_check(const size_t& now_step, const double& e1_single, const double& e2_single)
{
	return  sqrt(pow(e1_single, 2) + pow(e2_single, 2));
}

__device__ double compute_e1_single(const size_t& now_step, const double& qq_now_single, const double& EE0)
{
	double tao = 0.0;
	double t1 = 0.5 * DX * (now_step + 1);
	return  (EE0 / (1.0 + TP_const)) * qq_now_single * sin(Omega1 * t1 + tao) -
		(EE0*TP_const / (1.0 + TP_const)) * qq_now_single * sin(Omega2 * t1 + 2 * tao);
}

__device__ double compute_e2_single(const size_t& now_step, const double& qq_now_single, const double& EE0)
{
	double tao = 0.0;
	double t1 = 0.5 * DX * (now_step + 1);
	return  (EE0 / (1.0 + TP_const)) * qq_now_single * cos(Omega1 * t1 + tao) +
		(EE0*TP_const / (1.0 + TP_const)) * qq_now_single * cos(Omega2 * t1 + 2 * tao);

}

__host__  double compute_ee0_by_index(const int index)
{
	double EE0 = 2.742*pow(10, 3)*sqrt(pow(10.0, (12.0 + double(index)*0.2)));
	EE0 = EE0 / (5.1421*(pow(10.0, 11.0)));
	return  EE0;
}