#include "hip/hip_runtime.h"
﻿
#include "../include/Laser.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Runge_Kutta.cuh"

#include <cmath>



__device__ double CalculationE1(const nucleus& first, const nucleus& second)
{
	//坐标平方和
	const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	//const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	//const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);


	return  0.5 * px_py_pz_squre_sum_first - 2.0 / sqrt(loc_squre_sum_first) +
		1.0 / loc_squre_sum_first * Q_constant * Q_constant / 4.0 / A_hardness *
		exp(A_hardness * (1.0 - pow((loc_squre_sum_first * px_py_pz_squre_sum_first /
			                            Q_constant * Q_constant), 2))) +
		1.0 / sqrt(distance_squre) / 2.0;
}

__device__ double CalculationE2(const nucleus& first, const nucleus& second)
{
	//坐标平方和
	//const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	//const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);


	return  0.5 * px_py_pz_squre_sum_second - 2.0 / sqrt(loc_squre_sum_second) +
		1.0 / loc_squre_sum_second * Q_constant * Q_constant / 4.0 / A_hardness *
		exp(A_hardness * (1.0 - pow((loc_squre_sum_second * px_py_pz_squre_sum_second /
			                            Q_constant * Q_constant), 2))) +
		1.0 / sqrt(distance_squre) / 2.0;
}

