#include "hip/hip_runtime.h"
﻿
#include "../include/Laser.cuh"
#include "../include/Sci_Constant.h"
#include "../include/Runge_Kutta.cuh"

#include <cmath>





__global__ void pre_second_step_E_forcheck(const double* E1, const double* E2, double* E_check)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		E_check[idx] = sqrt(pow(E1[idx], 2) + pow(E2[idx], 2));
	}
}








__global__ void pre_second_step_e1(const double* QQ, const double EE0, double* E1)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		double t1 = 0.5 * DX * idx;
		/*hiprandStatePhilox4_32_10_t s;
		hiprand_init(idx, 0, 0, &s);
		double random = hiprand_uniform_double(&s);
		double tao = 2.0 * random * PI;*/
		double tao = 0.0;
		E1[idx] = (EE0 / (1.0 + TP_const)) * QQ[idx] * sin(Omega1 * t1 + tao) -
			(EE0*TP_const / (1.0 + TP_const)) * QQ[idx] * sin(Omega2 * t1 + 2 * tao);

	}

}


__global__ void pre_second_step_e2(const double* QQ, const double EE0, double* E2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		double t1 = 0.5 * DX * idx;
		/*	hiprandStatePhilox4_32_10_t s;
		hiprand_init(idx, 0, 0, &s);
		double random = hiprand_uniform_double(&s);
		double tao = 2.0 * random * PI;*/
		double tao = 0.0;

		E2[idx] = (EE0 / (1.0 + TP_const)) * QQ[idx] * cos(Omega1 * t1 + tao) +
			(EE0*TP_const / (1.0 + TP_const)) * QQ[idx] * cos(Omega2 * t1 + 2 * tao);

	}

}
















__device__ double CalculationE1(const particle& first, const particle& second)
{
	//坐标平方和
	const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	//const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	//const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);


	return  0.5 * px_py_pz_squre_sum_first - 2.0 / sqrt(loc_squre_sum_first) +
		1.0 / loc_squre_sum_first * Q_constant * Q_constant / 4.0 / A_hardness *
		exp(A_hardness * (1.0 - pow((loc_squre_sum_first * px_py_pz_squre_sum_first /
			                            Q_constant * Q_constant), 2))) +
		1.0 / sqrt(distance_squre) / 2.0;
}

__device__ double CalculationE2(const particle& first, const particle& second)
{
	//坐标平方和
	//const double loc_squre_sum_first = pow(first.x, 2) + pow(first.y, 2) + pow(first.z, 2);
	const double loc_squre_sum_second = pow(second.x, 2) + pow(second.y, 2) + pow(second.z, 2);
	//一阶导平方和
	//const double px_py_pz_squre_sum_first = pow(first.px, 2) + pow(first.py, 2) + pow(first.pz, 2);
	const double px_py_pz_squre_sum_second = pow(second.px, 2) + pow(second.py, 2) + pow(second.pz, 2);
	//两核距离平方
	const double distance_squre = nucleus_distance(first, second);


	return  0.5 * px_py_pz_squre_sum_second - 2.0 / sqrt(loc_squre_sum_second) +
		1.0 / loc_squre_sum_second * Q_constant * Q_constant / 4.0 / A_hardness *
		exp(A_hardness * (1.0 - pow((loc_squre_sum_second * px_py_pz_squre_sum_second /
			                            Q_constant * Q_constant), 2))) +
		1.0 / sqrt(distance_squre) / 2.0;
}

