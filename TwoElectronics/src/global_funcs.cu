#include "hip/hip_runtime.h"
﻿#pragma comment(lib, "cudart.lib")
#pragma comment(lib, "hiprand.lib")

#include "../include/global_funcs.h"
#include "../include/sci_const.h"
#include "../include/device_compute_funcs.cuh"
#include "../include/common.hpp"
#include "../include/PrintStruct.h"

#include ""
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <thr/xthrcommon.h>

//生成双精度01均匀分布随机数
//参数:	Array:双精度数组	Size:数组长度
//void UniformRandomArrayD(double* Array, const long Size)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateUniformDouble(gen, Array, Size);					//生成0-1均匀分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}
//
////生成双精度正态分布随机数
////参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Stddev:方差(0.7)
//void NormalRandomArrayD(double* Array, const long Size, double Mean, double Stddev)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Stddev);		//生成正态分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}


//准备矩阵，计算一次即可
//


__global__ void pre_step_init(nuclei* Array, const long size,
				const double min_r,const double min_p)
{
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{

		double2 two_rand;
		double4 four_rand;
		get_six_random(two_rand, four_rand, idx);

		double theta1 = two_rand.x * 2.0 * PI;
		double phi1 = two_rand.y * PI;


		Array[idx].first.x = min_r * sin(phi1) * cos(theta1);
		Array[idx].first.y = min_r * sin(phi1) * sin(theta1);
		Array[idx].first.z = min_r * cos(phi1);

		Array[idx].second.x = -Array[idx].first.x;
		Array[idx].second.y = -Array[idx].first.y;
		Array[idx].second.z = -Array[idx].first.z;


		double phi2 = four_rand.x * PI;
		double phi3 = four_rand.y * PI;
		double theta2 = four_rand.z * 2.0 * PI;
		double theta3 = four_rand.w * 2.0 * PI;

		Array[idx].first.px = min_p * cos(theta2)*sin(phi2);
		Array[idx].first.py = min_p * sin(theta2)*sin(phi2);
		Array[idx].first.pz = min_p * cos(phi2);

		Array[idx].second.px = min_p * cos(theta3)*sin(phi3);
		Array[idx].second.py = min_p * sin(theta3)*sin(phi3);
		Array[idx].second.pz = min_p * cos(phi3);
	}
	return;
}

__global__ void first_step_on_gpu(nuclei* first_arr, const long size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("%p\n", &first_arr);
	if(idx<size)
	{
		//printf("%d\n", idx);
		for (int i = 0; i < one_steps; i++)
			update_step_one(first_arr[idx].first, first_arr[idx].second);
	}
	
	
}


__global__ void pre_second_step_qq(double* QQ)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2 * two_steps)
	{

		double t1 = 0.5 * DX * (idx+1);
		QQ[idx] = pow((sin(Omega1 / 2.0 / (2 * N1_const + N2_const)*t1)), 2);
		//QQ[idx] = t1;
	}
	
}


__global__ void pre_second_step_E_forcheck(const double* E1,const double* E2,double* E_check)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2 * two_steps)
	{
		E_check[idx] = sqrt(pow(E1[idx], 2) + pow(E2[idx] ,2));
	}
}








__global__ void pre_second_step_e1(const double* QQ,const double EE0,double* E1)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2*two_steps)
	{
		double t1 = 0.5 * DX * idx;
		/*hiprandStatePhilox4_32_10_t s;
		hiprand_init(idx, 0, 0, &s);
		double random = hiprand_uniform_double(&s);
		double tao = 2.0 * random * PI;*/
		double tao = 0.0;
		E1[idx] = (EE0 / (1.0 + TP_const)) * QQ[idx] * sin(Omega1 * t1 + tao) -
			(EE0*TP_const / (1.0 + TP_const)) * QQ[idx] * sin(Omega2 * t1 + 2 * tao);

	}

}


__global__ void pre_second_step_e2(const double* QQ, const double EE0, double* E2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		double t1 = 0.5 * DX * idx;
	/*	hiprandStatePhilox4_32_10_t s;
		hiprand_init(idx, 0, 0, &s);
		double random = hiprand_uniform_double(&s);
		double tao = 2.0 * random * PI;*/
		double tao = 0.0;

		E2[idx] = (EE0 / (1.0 + TP_const)) * QQ[idx] * cos(Omega1 * t1 + tao) +
			(EE0*TP_const / (1.0 + TP_const)) * QQ[idx] * cos(Omega2 * t1 + 2 * tao);

	}

}









__global__ void second_step_on_gpu(nuclei* second_arr , const long size, double* E1,double* E2)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	double e1_laser_t1 = 0.0, e1_laser_t2 = 0.0, e1_laser_t3 = 0.0, e1_laser_t4 = 0.0;
	double e2_laser_t1 = 0.0, e2_laser_t2 = 0.0, e2_laser_t3 = 0.0, e2_laser_t4 = 0.0;
	int idx_of_ds=-1; // 相当于nn
	double t1 = 0.0, t2 = 0.0, t3 = 0.0, t4 = 0.0;
	double now_t = 0.0; //当前时间，相当于t(1)
	if (idx<size)
	{
		for (int i = 0; i < two_steps; i++)
		{
			//第一个激光场强度
			t1 = now_t;
			if (t1 == 0)
			{
				e1_laser_t1 = 0.0;
				e2_laser_t1 = 0.0;
			}
			else
			{
				idx_of_ds = (2.0 * t1) / DX - 1;
				e1_laser_t1 = E1[idx_of_ds];
				e2_laser_t1 = E2[idx_of_ds];
			}
			//第二个激光场强度
			t2 = now_t + DX / 2.0;
			idx_of_ds = 2.0 * t2 / DX- 1;
			e1_laser_t2 = E1[idx_of_ds];
			e2_laser_t2 = E2[idx_of_ds];
			//第三个激光场强度
			t3 = now_t + DX / 2.0;
			idx_of_ds = 2 * t3 / DX- 1;
			e1_laser_t3 = E1[idx_of_ds];
			e2_laser_t3 = E2[idx_of_ds];
			//第四个激光场强度
			t4 = now_t + DX;
			idx_of_ds = 2.0 * t4 / DX - 1;
			e1_laser_t4 = E1[idx_of_ds];
			e2_laser_t4 = E2[idx_of_ds];
			double4 e1_laser = make_double4(e1_laser_t1, e1_laser_t2, e1_laser_t3, e1_laser_t4);
			double4 e2_laser = make_double4(e2_laser_t1, e2_laser_t2, e2_laser_t3, e2_laser_t4);
			update_step_two(second_arr[idx].first, second_arr[idx].second,
							e1_laser,e2_laser);
			now_t = now_t + DX;
			/*if(idx_of_ds == -1 )
				update_step_two(second_arr[idx].first, second_arr[idx].second,
									0.0,DS[0],DS[0],DS[1]);
			else
			{
				update_step_two(second_arr[idx].first, second_arr[idx].second,
					DS[idx_of_ds], DS[idx_of_ds + 1], DS[idx_of_ds + 1], DS[idx_of_ds + 2]);
			}
			idx_of_ds += 2;*/

		}
		
			
	}
}





__global__ void second_step_on_gpu_fliter
(const nuclei* second_arr, nuclei* second__arr_filter,
	const long size, unsigned long long* count_z, unsigned long long* count_zz)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < size)
	{
		double ee1 = CalculationE1(second_arr[idx].first, second_arr[idx].second);
		double ee2 = CalculationE2(second_arr[idx].first, second_arr[idx].second);

		if(ee1*ee2 < 0 )
		{
			atomicAdd(count_z, 1);
		}
		if( (ee1 > 0) && (ee2 > 0))
		{
			unsigned long long temp_idx = atomicAdd(count_zz, 1);
			/*nuclei temp;
			temp.first = second_arr[idx].first;
			temp.second = second_arr[idx].second;
			second__arr_filter[temp_idx - 1] = temp;*/
		}
	}
	
}



/*
 * double ee1 = CalculationE1(second_arr[idx].first, second_arr[idx].second);
		double ee2 = CalculationE2(second_arr[idx].first, second_arr[idx].second);
		if (ee1>0 && ee2>0)
		{
			
			unsigned long long temp_idx = atomicAdd(ee1_ee2_count, 1);
			nuclei temp;
			temp.first = second_arr[idx].first;
			temp.second = second_arr[idx].second;
			second_arr_fliter[temp_idx-1] = temp;
		}
 */


void get_min_r_min_p(int nx, int ny, double& min_r, double& min_p)
{
	double *R_Arr = (double*)malloc(nx * sizeof(double));
	double *P_Arr = (double*)malloc(ny * sizeof(double));

	for (int i = 0; i < nx; i++)
		R_Arr[i] = 0.5 + 0.01 * i;
	for (int i = 0; i < ny; i++)
		P_Arr[i] = 0.0 + 0.01*i;


	double** mat = new double*[nx];
	for (int i = 0; i<nx; i++)
		mat[i] = new double[ny];

	double Vh, Vk, Ek;
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			Vh = pow(Q_constant, 2) / (4.0*A_hardness*pow(R_Arr[i], 2)) *
				exp(A_hardness * (1.0 - pow((R_Arr[i] * P_Arr[j] / Q_constant), 4)));
			Vk = -2.0 / R_Arr[i];
			Ek = P_Arr[j] * P_Arr[j] / 2.0;
			mat[i][j] = Vh + Vk + Ek + 1.065;
		}
	}

	int min_x_index, min_y_index;
	double min = mat[0][0];
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			if (min > mat[i][j])
			{
				min = mat[i][j];
				min_x_index = i;
				min_y_index = j;
			}
		}
	}

	min_r = R_Arr[min_x_index];
	min_p = P_Arr[min_y_index];

	return;
}





//用于双核粒子的随机数化  初始化
void NucleiPreRandom(nuclei* Array, const long size)
{
	//计算最小 r p;
	double min_r, min_p;
	get_min_r_min_p(NX_const, NY_const, min_r, min_p);
	
	
	int dimx = 256;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	pre_step_init <<< grid, block >>> (Array, size,min_r,min_p);
	CHECK(hipDeviceSynchronize());
}


void NucleiFisrtStep(nuclei* first_array, const long size)
{
	int dimx = 32;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	first_step_on_gpu <<< grid, block >>> (first_array, size);
	
	
	CHECK( hipGetLastError() );
	CHECK(hipDeviceSynchronize());
	
}



void NucleiSecondStepPreQQ(double* QQ)
{
	//准备矢量势
	int pre_dimx = 512;
	dim3 pre_block(pre_dimx);
	dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
	pre_second_step_qq <<< pre_grid, pre_block >>> (QQ);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	

}

void NucleiSecondStepPreECheck(const double* QQ,const double EE0, double* E_check)
{
	//申请当前激光场 E1 E2 空间
	double *host_e1, *host_e2;
	double *gpu_e1, *gpu_e2;
	long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
	CHECK(hipMalloc((void **)(&gpu_e1), bytes_of_e_laser));
	CHECK(hipMalloc((void **)(&gpu_e2), bytes_of_e_laser));
	host_e1 = (double*)malloc(bytes_of_e_laser);
	host_e2 = (double*)malloc(bytes_of_e_laser);

	//准备矢量势
	int pre_dimx = 512;
	dim3 pre_block(pre_dimx);
	dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
	pre_second_step_e1 << < pre_grid, pre_block >> > (QQ, EE0, gpu_e1);
	pre_second_step_e2 << < pre_grid, pre_block >> > (QQ, EE0, gpu_e2);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());

	
	PrintArray(host_e1, 2 * two_steps_in_host, "E1_check", 1);
	PrintArray(host_e2, 2 * two_steps_in_host, "E2_check", 1);
	

	pre_second_step_E_forcheck <<< pre_grid, pre_block >>> (gpu_e1,gpu_e2,E_check);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}




void NucleiSecondStepWholeLaserNoStream(nuclei* first_array, const long size, double* QQ)
{
	int n_streams = 1;


	unsigned long long *host_count_z_arr, *host_count_zz_arr;
	unsigned long long *gpu_count_z_arr, *gpu_count_zz_arr;
	const int size_ull = sizeof(unsigned long long);

	CHECK(hipMalloc((void**)&gpu_count_z_arr, n_streams * size_ull));
	CHECK(hipMalloc((void**)&gpu_count_zz_arr, n_streams * size_ull));

	//在CPU上分配页锁定内存  
	CHECK(hipHostAlloc((void**)&host_count_z_arr, n_streams * size_ull, hipHostMallocDefault));
	CHECK(hipHostAlloc((void**)&host_count_zz_arr, n_streams * size_ull, hipHostMallocDefault));
	for (int stream_index = 0; stream_index < n_streams; stream_index++)
	{
		double *gpu_e1, *gpu_e2;
		long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
		CHECK(hipMalloc((void **)(&gpu_e1), bytes_of_e_laser));
		CHECK(hipMalloc((void **)(&gpu_e2), bytes_of_e_laser));

		double EE0 = 2.742*pow(10, 3)*sqrt(pow(10.0, (12.0 + double(stream_index)*0.2)));
		EE0 = EE0 / (5.1421*(pow(10.0, 11.0)));

		int pre_dimx = 512;
		dim3 pre_block(pre_dimx);
		dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
		pre_second_step_e1 << < pre_grid, pre_block, 0, 0 >> > (QQ, EE0, gpu_e1);
		pre_second_step_e2 << < pre_grid, pre_block, 0, 0 >> > (QQ, EE0, gpu_e2);


		//计算第二步 一个激光场
		int dimx = 32;
		dim3 block(dimx);
		dim3 grid((size + block.x - 1) / block.x, 1);

		nuclei* gpu_second_arr_once, *gpu_second_filter_once;

		long long nBytes = size * sizeof(nuclei);
		CHECK(hipMalloc((void **)(&gpu_second_arr_once), nBytes));
		CHECK(hipMalloc((void **)(&gpu_second_filter_once), nBytes));

		CHECK(hipMemcpy(gpu_second_arr_once, first_array, nBytes, hipMemcpyDeviceToDevice));

		second_step_on_gpu << < grid, block, 0, 0 >> > (gpu_second_arr_once, size, gpu_e1, gpu_e2);

		second_step_on_gpu_fliter << < grid, block, 0, 0 >> > (gpu_second_arr_once,
			gpu_second_filter_once, size, gpu_count_z_arr + stream_index, gpu_count_zz_arr + stream_index);

		CHECK(hipMemcpy(host_count_z_arr + stream_index * size_ull,
			gpu_count_z_arr + size_ull * (stream_index),
			size_ull, hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(host_count_zz_arr + size_ull * (stream_index),
			gpu_count_zz_arr + size_ull * (stream_index),
			size_ull, hipMemcpyDeviceToHost));




		//printf("第一列z,第二列zz");
		printf("%.10f\t", EE0);
		printf("z: %lld \t", host_count_z_arr[stream_index]);
		printf("zz: %lld \n", host_count_zz_arr[stream_index]);
		CHECK(hipGetLastError());
	}
	CHECK(hipGetLastError());
	//CHECK(hipDeviceSynchronize());
}








void NucleiSecondStepWholeLaser(nuclei* first_array, const long size, double* QQ)
{
	int n_streams = 1;
	hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));

	for (int i = 0; i < n_streams; i++)
	{
		CHECK(hipStreamCreate(&(streams[i])));
	}

	unsigned long long *host_count_z_arr, *host_count_zz_arr;
	unsigned long long *gpu_count_z_arr, *gpu_count_zz_arr;
	const int size_ull = sizeof(unsigned long long);
	
	CHECK(hipMalloc((void**)&gpu_count_z_arr, n_streams * size_ull));
	CHECK(hipMalloc((void**)&gpu_count_zz_arr, n_streams * size_ull));

	//在CPU上分配页锁定内存  
	CHECK(hipHostAlloc((void**)&host_count_z_arr, n_streams * size_ull, hipHostMallocDefault));
	CHECK(hipHostAlloc((void**)&host_count_zz_arr, n_streams * size_ull, hipHostMallocDefault));
	for(int stream_index = 0 ; stream_index < n_streams ; stream_index++)
	{
		double *gpu_e1, *gpu_e2;
		long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
		CHECK(hipMalloc((void **)(&gpu_e1), bytes_of_e_laser));
		CHECK(hipMalloc((void **)(&gpu_e2), bytes_of_e_laser));
		
		double EE0 = 2.742*pow(10, 3)*sqrt(pow(10.0, (12.0 + double(stream_index)*0.2)));
		EE0 = EE0 / (5.1421*(pow(10.0, 11.0)));
				
		int pre_dimx = 512;
		dim3 pre_block(pre_dimx);
		dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
		pre_second_step_e1 <<< pre_grid, pre_block ,0, streams[stream_index]>>> (QQ, EE0, gpu_e1);
		pre_second_step_e2 <<< pre_grid, pre_block , 0, streams[stream_index] >>> (QQ, EE0, gpu_e2);
		

		//计算第二步 一个激光场
		int dimx = 32;
		dim3 block(dimx);
		dim3 grid((size + block.x - 1) / block.x, 1);

		nuclei* gpu_second_arr_once,*gpu_second_filter_once;
		
		long long nBytes = size * sizeof(nuclei);
		CHECK(hipMalloc((void **)(&gpu_second_arr_once), nBytes));
		CHECK(hipMalloc((void **)(&gpu_second_filter_once), nBytes));

		CHECK(hipMemcpy(gpu_second_arr_once, first_array, nBytes, hipMemcpyDeviceToDevice));
		
		second_step_on_gpu <<< grid, block, 0, streams[stream_index] >>> (gpu_second_arr_once, size, gpu_e1, gpu_e2);

		second_step_on_gpu_fliter <<< grid, block, 0, streams[stream_index] >>> (gpu_second_arr_once, 
			gpu_second_filter_once, size, gpu_count_z_arr + stream_index , gpu_count_zz_arr + stream_index);
		
		hipMemcpyAsync(host_count_z_arr + stream_index * size_ull,
			gpu_count_z_arr + size_ull * (stream_index),
			size_ull, hipMemcpyDeviceToHost, streams[stream_index]);
		hipMemcpyAsync(host_count_zz_arr + size_ull * (stream_index),
			gpu_count_zz_arr + size_ull * (stream_index),
			size_ull, hipMemcpyDeviceToHost, streams[stream_index]);
	
		//printf("第一列z,第二列zz");
		printf("%.10f\t", EE0);
		printf("z: %lld \t", host_count_z_arr[stream_index]);
		printf("zz: %lld \n", host_count_zz_arr[stream_index]);
		CHECK(hipGetLastError());
	}
	CHECK(hipGetLastError());
	//CHECK(hipDeviceSynchronize());
	for (int i = 0; i < n_streams; i++)
	{
		CHECK(hipStreamDestroy(streams[i]));
		
	}
	



	free(streams);
}








void compute_on_gpu_one(const long pairs,const char* file_name)
{
	long long nBytes = pairs * sizeof(nuclei);
	printf("Use %lld Bytes %lfMB\n", nBytes, nBytes / double(1024 * 1024));
	nuclei *gpu_init,*gpu_first,*gpu_second,*gpu_second_fliter;
	nuclei *host_init,*host_first,*host_second,*host_second_fliter;
	host_init = (nuclei*)malloc(nBytes);
	host_first = (nuclei*)malloc(nBytes);
	host_second = (nuclei*)malloc(nBytes);
	host_second_fliter = (nuclei*)malloc(nBytes);


	//初始化！
	//申请init空间
	double start = seconds();
	CHECK(hipMalloc((void **)(&gpu_init), nBytes));
	//计算
	NucleiPreRandom(gpu_init, pairs);

	//把值赋给第一步(也申请了第一步的空间)
	CHECK(hipMalloc((void **)(&gpu_first), nBytes));
	CHECK(hipMemcpy(gpu_first, gpu_init, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_init, gpu_init, nBytes, hipMemcpyDeviceToHost));
	
	PrintStruct(host_init, pairs, file_name, 0);
	//释放init空间
	CHECK(hipFree(gpu_init));
	double elapse = seconds();
	printf("Inition compltete %lf\n", elapse - start);
	//初始化完成！


	 //第一步计算
	//first空间在之前申请过了
	 start = seconds();
	//计算
	NucleiFisrtStep(gpu_first, pairs);
	

	//把值赋给第二步(也申请了第二步的空间)
	CHECK(hipMalloc((void **)(&gpu_second), nBytes));
	CHECK(hipMemcpy(gpu_second, gpu_first, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_first, gpu_first, nBytes, hipMemcpyDeviceToHost));
	PrintStruct(host_first, pairs, file_name, 1);
	//释放first空间
	CHECK(hipFree(gpu_first));
	elapse = seconds();
	printf("FirstStep compltete %lf\n", elapse - start);
	//第一步完成！


	//第二步计算
	start = seconds();
	//准备导数
	double *gpu_qq;
	double *host_qq;
	long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
	CHECK(hipMalloc((void **)(&gpu_qq), bytes_of_e_laser));
	host_qq = (double*)malloc(bytes_of_e_laser);
	NucleiSecondStepPreQQ(gpu_qq);
	CHECK(hipMemcpy(host_qq, gpu_qq, bytes_of_e_laser, hipMemcpyDeviceToHost));
	//验证QQ 通过！
	//PrintArray(host_qq, 2 * two_steps_in_host, "QQ_Check", 0);


	////测试E通过！
	//double *gpu_e_check, *host_e_check;
	//CHECK(hipMalloc((void **)(&gpu_e_check), bytes_of_e_laser));
	//host_e_check = (double*)malloc(bytes_of_e_laser);
	//double EE0 = sqrt(1e15 / 3.51e16);
	//NucleiSecondStepPreECheck(gpu_qq, EE0, gpu_e_check);
	//CHECK(hipMemcpy(host_e_check, gpu_e_check, bytes_of_e_laser, hipMemcpyDeviceToHost));
	//PrintArray(host_e_check, 2 * two_steps_in_host, "e_check", 0);
 
	NucleiSecondStepWholeLaser(gpu_second, pairs, gpu_qq);




	////计算

	//NucleiSecondStep(gpu_second, gpu_second_fliter,pairs, gpu_aw, gpu_ds,gpu_count);

	////拷回并保存
	//CHECK(hipMemcpy(host_second, gpu_second, nBytes, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_second_fliter, gpu_second_fliter, nBytes, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_aw, gpu_aw, bytes_of_aw_ds, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_ds, gpu_ds, bytes_of_aw_ds, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_count, gpu_count, bytes_of_u_long, hipMemcpyDeviceToHost));
	//printf("%ld\n", *host_count);

	//PrintStruct(host_second, pairs,file_name , 2);
	//PrintArray(host_aw, 2 * two_steps_in_host, file_name, 0);
	//PrintArray(host_ds, 2 * two_steps_in_host, file_name, 1);
	//PrintStruct(host_second_fliter, *host_count, file_name, 3);
	////释放second空间
	//CHECK(hipFree(gpu_second));
	//CHECK(hipFree(gpu_aw));
	//CHECK(hipFree(gpu_ds));
	//
	elapse = seconds();
	printf("SecondStep compltete %lf\n", elapse - start);
	// 第二步完成！

	//释放主机内存空间
	//free(host_aw);
	//free(host_ds);
	//free(host_first);
	//free(host_init);
	//free(host_init);




	return;
}