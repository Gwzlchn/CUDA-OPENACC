#include "hip/hip_runtime.h"
﻿#pragma comment(lib, "cudart.lib")
#pragma comment(lib, "hiprand.lib")

#include "../include/global_funcs.h"
#include "../include/sci_const.h"
#include "../include/device_compute_funcs.cuh"
#include "../include/common.hpp"
#include "../include/PrintStruct.h"

#include ""
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

//生成双精度01均匀分布随机数
//参数:	Array:双精度数组	Size:数组长度
//void UniformRandomArrayD(double* Array, const long Size)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateUniformDouble(gen, Array, Size);					//生成0-1均匀分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}
//
////生成双精度正态分布随机数
////参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Stddev:方差(0.7)
//void NormalRandomArrayD(double* Array, const long Size, double Mean, double Stddev)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Stddev);		//生成正态分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}


//准备矩阵，计算一次即可
//


__global__ void pre_step_init(nuclei* Array, const long& size,
				const double& min_r,const double& min_p)
{
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{
		distribution(Array[idx].first, Array[idx].second, idx, min_r, min_p);
	}
	return;
}

__global__ void first_step_on_gpu(nuclei* first_arr, const long size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("%p\n", &first_arr);
	if(idx<size)
	{
		//printf("%d\n", idx);
		for (int i = 0; i < one_steps; i++)
			update_step_one(first_arr[idx].first, first_arr[idx].second);
	}
	
	
}


__global__ void pre_second_step_qq(double* QQ)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2 * two_steps)
	{

		double t1 = 0.5 * DX * idx;
		QQ[idx] = pow((sin(Omega1 / 2.0 / (2 * N1_const + N2_const))*t1), 2);
	    
		
	}
	
}


__global__ void pre_second_step_E_forcheck(const double* E1,const double* E2,double* E_check)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2 * two_steps)
	{

		double t1 = 0.5 * DX * idx;
		E_check[idx] = sqrt(pow(E1[idx], 2) + pow(E2[idx] ,2));
	    
		
	}
}








__global__ void pre_second_step_e1(const double* QQ,const double EE0,double* E1)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2*two_steps)
	{
		double t1 = 0.5 * DX * idx;
		hiprandStatePhilox4_32_10_t s;
		hiprand_init(idx, 0, 0, &s);
		double random = hiprand_uniform_double(&s);
		double tao = 2.0 * random * PI;

		E1[idx] = (EE0 / (1.0 + TP_const)) * QQ[idx] * sin(Omega1 * t1 + tao) -
			(EE0*TP_const / (1.0 + TP_const)) * QQ[idx] * sin(Omega2 * t1 + 2 * tao);

	}

}


__global__ void pre_second_step_e2(const double* QQ, const double EE0, double* E2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < 2 * two_steps)
	{
		double t1 = 0.5 * DX * idx;
		hiprandStatePhilox4_32_10_t s;
		hiprand_init(idx, 0, 0, &s);
		double random = hiprand_uniform_double(&s);
		double tao = 2.0 * random * PI;

		E2[idx] = (EE0 / (1.0 + TP_const)) * QQ[idx] * sin(Omega1 * t1 + tao) +
			(EE0*TP_const / (1.0 + TP_const)) * QQ[idx] * sin(Omega2 * t1 + 2 * tao);

	}

}









__global__ void second_step_on_gpu(nuclei* second_arr , const long size, double* E1,double* E2)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	double e1_laser_t1=0.0, e1_laser_t2=0.0, e1_laser_t3=0.0, e1_laser_t4=0.0;
	double e2_laser_t1 = 0.0, e2_laser_t2 = 0.0, e2_laser_t3 = 0.0, e2_laser_t4 = 0.0;
	int idx_of_ds=-1; // 相当于nn
	double t1=0.0, t2=0.0, t3=0.0, t4=0.0;
	double now_t=0.0; //当前时间，相当于t(1)
	if (idx<size)
	{
		for (int i = 0; i < two_steps; i++)
		{
			//第一个激光场强度
			t1 = now_t;
			if (t1 == 0)
				e1_laser_t1 = 0.0;
			else
			{
				idx_of_ds = (2.0 * t1) / DX - 1;
				e1_laser_t1 = E1[idx_of_ds];
				e2_laser_t1 = E2[idx_of_ds];
			}
			//第二个激光场强度
			t2 = now_t + DX / 2.0;
			idx_of_ds = 2.0 * t2 / DX- 1;
			e1_laser_t2 = E1[idx_of_ds];
			e2_laser_t2 = E2[idx_of_ds];
			//第三个激光场强度
			t3 = now_t + DX / 2.0;
			idx_of_ds = 2 * t3 / DX- 1;
			e1_laser_t3 = E1[idx_of_ds];
			e2_laser_t3 = E2[idx_of_ds];
			//第四个激光场强度
			t4 = now_t + DX;
			idx_of_ds = 2.0 * t4 / DX - 1;
			e1_laser_t4 = E1[idx_of_ds];
			e2_laser_t4 = E2[idx_of_ds];
			double4 e1_laser = make_double4(e1_laser_t1, e2_laser_t2, e1_laser_t3, e1_laser_t4);
			double4 e2_laser = make_double4(e2_laser_t1, e2_laser_t2, e2_laser_t3, e2_laser_t4);
			update_step_two(second_arr[idx].first, second_arr[idx].second,
							e1_laser,e2_laser);
			now_t = now_t + DX;
			/*if(idx_of_ds == -1 )
				update_step_two(second_arr[idx].first, second_arr[idx].second,
									0.0,DS[0],DS[0],DS[1]);
			else
			{
				update_step_two(second_arr[idx].first, second_arr[idx].second,
					DS[idx_of_ds], DS[idx_of_ds + 1], DS[idx_of_ds + 1], DS[idx_of_ds + 2]);
			}
			idx_of_ds += 2;*/

		}
		
			
	}
}


/*
 * double ee1 = CalculationE1(second_arr[idx].first, second_arr[idx].second);
		double ee2 = CalculationE2(second_arr[idx].first, second_arr[idx].second);
		if (ee1>0 && ee2>0)
		{
			
			unsigned long long temp_idx = atomicAdd(ee1_ee2_count, 1);
			nuclei temp;
			temp.first = second_arr[idx].first;
			temp.second = second_arr[idx].second;
			second_arr_fliter[temp_idx-1] = temp;
		}
 */


void get_min_r_min_p(int nx, int ny, double& min_r, double& min_p)
{
	double *R_Arr = (double*)malloc(nx * sizeof(double));
	double *P_Arr = (double*)malloc(ny * sizeof(double));

	for (int i = 0; i < nx; i++)
		R_Arr[i] = 0.5 + 0.01 * i;
	for (int i = 0; i < ny; i++)
		P_Arr[i] = 0.0 + 0.01*i;


	double** mat = new double*[nx];
	for (int i = 0; i<nx; i++)
		mat[i] = new double[ny];

	double Vh, Vk, Ek;
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			Vh = pow(Q_constant, 2) / (4.0*A_hardness*pow(R_Arr[i], 2)) *
				exp(A_hardness * (1.0 - pow((R_Arr[i] * P_Arr[j] / Q_constant), 4)));
			Vk = -2.0 / R_Arr[i];
			Ek = P_Arr[j] * P_Arr[j] / 2.0;
			mat[i][j] = Vh + Vk + Ek + 1.065;
		}
	}

	int min_x_index, min_y_index;
	double min = mat[0][0];
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			if (min > mat[i][j])
			{
				min = mat[i][j];
				min_x_index = i;
				min_y_index = j;
			}
		}
	}

	min_r = R_Arr[min_x_index];
	min_p = P_Arr[min_y_index];

	return;
}





//用于双核粒子的随机数化  初始化
void NucleiPreRandom(nuclei* Array, const long size)
{
	//计算最小 r p;
	double min_r, min_p;
	get_min_r_min_p(NX_const, NY_const, min_r, min_p);
	
	
	int dimx = 256;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	pre_step_init <<< grid, block >>> (Array, size,min_r,min_p);
	CHECK(hipDeviceSynchronize());
}


void NucleiFisrtStep(nuclei* first_array, const long size)
{
	int dimx = 32;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	first_step_on_gpu <<< grid, block >>> (first_array, size);
	
	
	CHECK( hipGetLastError() );
	CHECK(hipDeviceSynchronize());
	
}



void NucleiSecondStepPreQQ(double* QQ)
{
	//准备矢量势
	int pre_dimx = 512;
	dim3 pre_block(pre_dimx);
	dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
	pre_second_step_qq <<< pre_grid, pre_block >>> (QQ);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

}

void NucleiSecondStepPreECheck(const double* QQ,const double EE0, double* E_check)
{
	//申请当前激光场 E1 E2 空间
	double *host_e1, *host_e2;
	double *gpu_e1, *gpu_e2;
	long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
	CHECK(hipMalloc((void **)(&gpu_e1), bytes_of_e_laser));
	CHECK(hipMalloc((void **)(&gpu_e2), bytes_of_e_laser));
	host_e1 = (double*)malloc(bytes_of_e_laser);
	host_e2 = (double*)malloc(bytes_of_e_laser);

	//准备矢量势
	int pre_dimx = 512;
	dim3 pre_block(pre_dimx);
	dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
	pre_second_step_e1 << < pre_grid, pre_block >> > (QQ, EE0, gpu_e1);
	pre_second_step_e2 << < pre_grid, pre_block >> > (QQ, EE0, gpu_e2);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());

	
	
	

	pre_second_step_E_forcheck <<< pre_grid, pre_block >>> (gpu_e1,gpu_e2,E_check);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}










void NucleiSecondStepOneLaser(nuclei* second_array , const long size,double* QQ,double EE0)
{
	//申请当前激光场 E1 E2 空间
	double *host_e1,*host_e2;
	double *gpu_e1, *gpu_e2;
	long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
	CHECK(hipMalloc((void **)(&gpu_e1), bytes_of_e_laser));
	CHECK(hipMalloc((void **)(&gpu_e2), bytes_of_e_laser));
	host_e1 = (double*)malloc(bytes_of_e_laser);
	host_e2 = (double*)malloc(bytes_of_e_laser);

	//准备矢量势
	int pre_dimx = 512;
	dim3 pre_block(pre_dimx);
	dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
	pre_second_step_e1 <<< pre_grid, pre_block >>> (QQ, EE0, gpu_e1);
	pre_second_step_e2 <<< pre_grid, pre_block >>> (QQ, EE0, gpu_e2);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());

	//计算第二步 一个激光场
 	int dimx = 32;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	second_step_on_gpu <<< grid, block >>> (second_array,size,gpu_e1,gpu_e2);
	
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
}



void compute_on_gpu_one(const long pairs,const char* file_name)
{
	long long nBytes = pairs * sizeof(nuclei);
	printf("Use %lld Bytes %lfMB\n", nBytes, nBytes / double(1024 * 1024));
	nuclei *gpu_init,*gpu_first,*gpu_second,*gpu_second_fliter;
	nuclei *host_init,*host_first,*host_second,*host_second_fliter;
	host_init = (nuclei*)malloc(nBytes);
	host_first = (nuclei*)malloc(nBytes);
	host_second = (nuclei*)malloc(nBytes);
	host_second_fliter = (nuclei*)malloc(nBytes);


	//初始化！
	//申请init空间
	double start = seconds();
	CHECK(hipMalloc((void **)(&gpu_init), nBytes));
	//计算
	NucleiPreRandom(gpu_init, pairs);

	//把值赋给第一步(也申请了第一步的空间)
	CHECK(hipMalloc((void **)(&gpu_first), nBytes));
	CHECK(hipMemcpy(gpu_first, gpu_init, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_init, gpu_init, nBytes, hipMemcpyDeviceToHost));
	
	PrintStruct(host_init, pairs, file_name, 0);
	//释放init空间
	CHECK(hipFree(gpu_init));
	double elapse = seconds();
	printf("Inition compltete %lf\n", elapse - start);
	//初始化完成！


	//第一步计算
	//first空间在之前申请过了
	 start = seconds();
	//计算
	NucleiFisrtStep(gpu_first, pairs);
	

	//把值赋给第二步(也申请了第二步的空间)
	CHECK(hipMalloc((void **)(&gpu_second), nBytes));
	CHECK(hipMemcpy(gpu_second, gpu_first, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_first, gpu_first, nBytes, hipMemcpyDeviceToHost));
	PrintStruct(host_first, pairs, file_name, 1);
	//释放first空间
	CHECK(hipFree(gpu_first));
	elapse = seconds();
	printf("FirstStep compltete %lf\n", elapse - start);
	//第一步完成！


	//第二步计算
	start = seconds();
	//准备导数
	double *gpu_qq;
	double *host_qq;
	long bytes_of_e_laser = sizeof(double) * 2 * two_steps_in_host;
	CHECK(hipMalloc((void **)(&gpu_qq), bytes_of_e_laser));
	host_qq = (double*)malloc(bytes_of_e_laser);
	NucleiSecondStepPreQQ(gpu_qq);

	double *gpu_e_check, *host_e_check;
	CHECK(hipMalloc((void **)(&gpu_e_check), bytes_of_e_laser));
	host_e_check = (double*)malloc(bytes_of_e_laser);
	double EE0 = sqrt(1e15 / 3.51e16);
	NucleiSecondStepPreECheck(gpu_qq, EE0, gpu_e_check);
	CHECK(hipMemcpy(host_e_check, gpu_e_check, bytes_of_e_laser, hipMemcpyDeviceToHost));
	PrintArray(host_e_check, 2 * two_steps_in_host, "e_check", 0);

	
	////电离率计数
	//unsigned long long*gpu_count,*host_count;
	//int bytes_of_u_long = sizeof(unsigned long long);
	//host_count = (unsigned long long*)malloc(bytes_of_u_long);
	//CHECK(hipMalloc((void **)(&gpu_count), bytes_of_u_long));
	//CHECK(hipMalloc((void **)(&gpu_second_fliter), nBytes));

	////检查E_check

	//



	////计算

	//NucleiSecondStep(gpu_second, gpu_second_fliter,pairs, gpu_aw, gpu_ds,gpu_count);

	////拷回并保存
	//CHECK(hipMemcpy(host_second, gpu_second, nBytes, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_second_fliter, gpu_second_fliter, nBytes, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_aw, gpu_aw, bytes_of_aw_ds, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_ds, gpu_ds, bytes_of_aw_ds, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(host_count, gpu_count, bytes_of_u_long, hipMemcpyDeviceToHost));
	//printf("%ld\n", *host_count);

	//PrintStruct(host_second, pairs,file_name , 2);
	//PrintArray(host_aw, 2 * two_steps_in_host, file_name, 0);
	//PrintArray(host_ds, 2 * two_steps_in_host, file_name, 1);
	//PrintStruct(host_second_fliter, *host_count, file_name, 3);
	////释放second空间
	//CHECK(hipFree(gpu_second));
	//CHECK(hipFree(gpu_aw));
	//CHECK(hipFree(gpu_ds));
	//
	//elapse = seconds();
	//printf("SecondStep compltete %lf\n", elapse - start);
	// 第二步完成！

	//释放主机内存空间
	//free(host_aw);
	//free(host_ds);
	//free(host_first);
	//free(host_init);
	//free(host_init);




	return;
}