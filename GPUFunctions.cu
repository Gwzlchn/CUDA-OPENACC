#include "hip/hip_runtime.h"
#include"GPUFunctions.h"
#include ""
#include "HostFunctions.hpp"
#include "common.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<math.h>


__device__ double fx(double x)
{

	return -(x/(pow(sqrt(pow(x,2.0)+pow(A,2.0)),3.0)));
}


__device__ double Ekall(double x)
{

	return E0+1.0/(sqrt(pow(x,2.0)+pow(A,2.0)));
}

__device__ double Px(double x)
{
	return sqrt(2*Ekall(x));
}

//数据初始化应该单独用一个kernel函数，计算fx px的初值
//待完成。mark一下
//1118wzl已完成

__global__ void InitialKernel(double* Result,int nx,int ny)
{
	//第一列已经是随机数了
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx ){
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
			//第二列为第一列各自的px初值，如果出现根号下小于零的情况，直接赋值0，计算部分判断简单些（nan判定很烦……）
            if((idx>=1*nx)&&(idx<2*nx)){
				if(Ekall(Result[idx-nx])>=0.0)
					Result[idx] = Px(double(Result[idx-nx]));
				else Result[idx] = 0.0;
			}
			//第三列为第一列各自的fx初值，出现小于零情况同理。
			if((idx>=2*nx)&&(idx<3*nx)){
				if(Result[idx-1*nx]>0.0)
					Result[idx] = fx(double(Result[idx-2*nx]));
				else Result[idx] = 0.0;
			}
			
			//第四五六列为前三列的复制，为了compute函数准备
			if((idx>=3*nx)&&(idx<4*nx)){
				if(Result[idx-2*nx]>0.0)
					Result[idx] = Result[idx-3*nx];
				else Result[idx] = 0.0;
			}
			if((idx>=4*nx)&&(idx<5*nx))
				Result[idx] = Result[idx-3*nx];
			
		}
	}

}


void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //生成随机数变量
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
    hiprandGenerateNormalDouble(gen, ip, size, 0, 0.7);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
    hiprandDestroyGenerator(gen);                         	//释放指针
	return;
	
	
}


void  InitialMatrix(double* d_Result,int nx,int ny){
	NormalRandom(d_Result,nx);
	//分配grid,block大小
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	InitialKernel<<<grid,block>>>(d_Result,nx,ny);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	
	
	
	//保存数据仅仅为了测试用，写好compute部分以后肯定不用保存这个数据了……
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	double *h_gpuRef;
	h_gpuRef = (double *)malloc(nBytes);
	CHECK(hipMemcpy(h_gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"init.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE InitialKernel DATA elapsed %lf sec\n",iElaps);
	
	
}






__device__ void updateXi(double& xi,double& pxi)
{
	//const double DX=0.027;
	double K1  = pxi;
	double K11 = fx(xi);
	
	double K2  = pxi + K11/2.0*DX;
	double K22 = fx(xi + K1/2.0*DX);
	
	double K3  = pxi + K22/2.0*DX;
	double K33 = fx(xi + K2/2.0*DX);
	
	double K4  = pxi+K33*DX;
	double K44 = fx(xi+K3*DX);
	
	xi  = xi  + DX * (K1  + 2*K2  + 2*K3  + K4)/6.0;
	pxi = pxi + DX * (K11 + K22*2 + K33*2 + K44)/6.0;
	return;
}










__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int idxOfXi  = 3 * nx + ix;
	unsigned int idxOfPxi = 4 * nx + ix;
	
	unsigned int Steps = TOSTOP/DX;
	
	
    if(ix<nx && Result[idxOfXi]!=0.0){
		for(int i=0;i<Steps;i++){
			updateXi(Result[idxOfXi],Result[idxOfPxi]);
			
		}
	}
}







 void ComputeOnGPU1(double* Result,int nx,int ny,double* h_gpuRef){
	
	
	//分配grid,block大小
	int dimx = 512;
    dim3 block(dimx);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	 CHECK(hipDeviceSynchronize());
	//如果核函数错误，返回信息
    CHECK(hipGetLastError());
	// GPU数据拷贝回主机
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	CHECK(hipMemcpy(h_gpuRef, Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"gpu.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE ComputeKernel DATA elapsed %lf sec\n",iElaps);
	return;
}









