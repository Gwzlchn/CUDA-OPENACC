#include "hip/hip_runtime.h"
#include"GPUFunctions.h"
#include ""
#include "HostFunctions.hpp"
#include "common.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<math.h>


__device__ double fx(double x)
{

	return -1.0/(pow(sqrt(pow(x,2.0)+pow(A,2.0)),3.0));
}


__device__ double Ekall(double x)
{

	return E0+1.0/(sqrt(pow(x,2.0)+pow(A,2.0)));
}

__device__ double Px(double x)
{
	return sqrt(2*Ekall(x));
}

//数据初始化应该单独用一个kernel函数，计算fx px的初值
//待完成。mark一下
//1118wzl已完成

__global__ void InitialKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx ){
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
            if((idx>=1*nx)&&(idx<2*nx)){
				if(Ekall(Result[idx-nx])>=0.0)
					Result[idx] = Px(double(Result[idx-nx]));
				else Result[idx] = 0.0;
			}
				
			if((idx>=2*nx)&&(idx<3*nx)){
				if(Result[idx-1*nx]>0.0)
					Result[idx] = fx(double(Result[idx-2*nx]));
				else Result[idx] = 0.0;
			}
				
			if((idx>=3*nx)&&(idx<4*nx)){
				if(Result[idx-2*nx]>0.0)
					Result[idx] = Result[idx-3*nx];
				else Result[idx] = 0.0;
			}
			if((idx>=4*nx)&&(idx<5*nx))
				Result[idx] = Result[idx-3*nx];
			if((idx>=5*nx)&&(idx<6*nx))
				Result[idx] = Result[idx-3*nx];
		}
	}

}


void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //生成随机数变量
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
    hiprandGenerateNormalDouble(gen, ip, size, 0, 1);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
    hiprandDestroyGenerator(gen);                         //释放指针
	return;
	
	
}


void  InitialMatrix(double* d_Result,int nx,int ny){
	NormalRandom(d_Result,nx);
	//分配grid,block大小
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	InitialKernel<<<grid,block>>>(d_Result,nx,ny);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	double *h_gpuRef;
	h_gpuRef = (double *)malloc(nBytes);
	CHECK(hipMemcpy(h_gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"init.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	
	
}






__device__ double updateXi(double xi,double pxi)
{
	double K1=pxi;
	double K2=xi+K1/2.0*DX;
	double K3=xi+K2/2.0*DX;
	double K4=xi+K3*DX;
	
	return xi+DX*(K1+2*K2+2*K3+K4)/6.0;
}


__device__ double updatePxi(double pxi,double fxi)
{
	double K1=fxi;
	double K2=pxi+K1/2.0*DX;
	double K3=pxi+K2/2.0*DX;
	double K4=pxi+K3*DX;
	
	return pxi+DX*(K1+2*K2+2*K3+K4)/6.0;
}





__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx ){
		
		return;
		
		}
}







 void ComputeOnGPU1(double* Result,int nx,int ny,double* h_gpuRef){
	
	
		//分配grid,block大小
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	 CHECK(hipDeviceSynchronize());
	    //如果核函数错误，返回信息
    CHECK(hipGetLastError());
	 // GPU数据拷贝回主机
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	CHECK(hipMemcpy(h_gpuRef, Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"gpu.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	return;
}









