#include "hip/hip_runtime.h"
#include"GPUFunctions.h"
#include ""
#include "HostFunctions.hpp"
#include "common.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<math.h>


__device__ double fx(double x)
{

	return -1.0/(pow(sqrt(pow(x,2.0)+pow(A,2.0)),3.0));
}


__device__ double Ekall(double x)
{

	return E0+1.0/(sqrt(pow(x,2.0)+pow(A,2.0)));
}

__device__ double Px(double x)
{
	return sqrt(2*Ekall(x));
}

//数据初始化应该单独用一个kernel函数，计算fx px的初值
//待完成。mark一下

__global__ void InitialKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx ){

        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
            
			if((idx>=1*nx)&&(idx<2*nx))
				Result[idx] = Px(double(Result[idx-nx]));
			if((idx>=2*nx)&&(idx<3*nx))
				Result[idx] = fx(double(Result[idx-2*nx]));
			
			
			if((idx>=3*nx)&&(idx<4*nx)){
				Result[idx] = Result[idx-3*nx];
			}
			if((idx>=4*nx)&&(idx<5*nx)){
				Result[idx] = Result[idx-3*nx];
			}
			
				
				/*const double dx=0.00001;
				int i,n=1+(2*PI)/dx;
				double temp;
				for (i = 1; i < n; i++){
					temp=rk4(dx, Result[idx-2*nx] + dx * (i - 1), Result[idx]);
					Result[idx] = temp;*/
					
		}
	}

}


void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //生成随机数变量
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
    hiprandGenerateNormalDouble(gen, ip, size, 0, 2);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
    hiprandDestroyGenerator(gen);                         //释放指针
	return;
	
	
}


void  InitialMatrix(double* d_Result,int nx,int ny,dim3 grid,dim3 block){
	NormalRandom(d_Result,nx);
	InitialKernel<<<grid,block>>>(d_Result,nx,ny);
	    CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	double *h_gpuRef;
	h_gpuRef = (double *)malloc(nBytes);
	CHECK(hipMemcpy(h_gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"init.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	
	
}






__device__ double updateXi(double xi,double pxi)
{
	double K1=pxi;
	double K2=xi+K1/2.0*DX;
	double K3=xi+K2/2.0*DX;
	double K4=xi+K3*DX;
	
	return xi+DX*(K1+2*K2+2*K3+K4)/6.0;
}


__device__ double updatePxi(double pxi,double fxi)
{
	double K1=fxi;
	double K2=pxi+K1/2.0*DX;
	double K3=pxi+K2/2.0*DX;
	double K4=pxi+K3*DX;
	
	return pxi+DX*(K1+2*K2+2*K3+K4)/6.0;
}





__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx ){

        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
            
			if((idx>=1*nx)&&(idx<2*nx))
				Result[idx] = Px(double(Result[idx-nx]));
			if((idx>=2*nx)&&(idx<3*nx))
				Result[idx] = fx(double(Result[idx-2*nx]));
			
			
			if((idx>=3*nx)&&(idx<4*nx)){
				Result[idx] = Result[idx-3*nx];
				int i,n=1+(TOSTOP)/DX;
				for(i=1;i<n;i++)
					Result[idx]=updateXi(Result[idx],DX);
			}
			
				
				/*const double dx=0.00001;
				int i,n=1+(2*PI)/dx;
				double temp;
				for (i = 1; i < n; i++){
					temp=rk4(dx, Result[idx-2*nx] + dx * (i - 1), Result[idx]);
					Result[idx] = temp;*/
					
		}
	}

}






 void ComputeOnGPU1(double* Result,int nx,int ny,dim3 grid,dim3 block,double* h_gpuRef){
	
	
	
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	 CHECK(hipDeviceSynchronize());
	    //如果核函数错误，返回信息
    CHECK(hipGetLastError());
	 // GPU数据拷贝回主机
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	CHECK(hipMemcpy(h_gpuRef, Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"gpu.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	return;
}









