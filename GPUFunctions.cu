#include "hip/hip_runtime.h"
#include"GPUFunctions.h"
#include ""
#include "HostFunctions.hpp"
#include "common.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<math.h>


__device__ double fx(double x)
{

	return -1.0/(pow(sqrt(pow(x,2.0)+pow(A,2.0)),3.0));
}


__device__ double Ekall(double x)
{

	return E0+1.0/(sqrt(pow(x,2.0)+pow(A,2.0)));
}

__device__ double Px(double x)
{
	return sqrt(2*Ekall(x));
}

//数据初始化应该单独用一个kernel函数，计算fx px的初值
//待完成。mark一下
//1118wzl已完成

__global__ void InitialKernel(double* Result,int nx,int ny)
{
	//第一列已经是随机数了
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx ){
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
			//第二列为第一列各自的px初值，如果出现根号下小于零的情况，直接赋值0，计算部分判断简单些（nan判定很烦……）
            if((idx>=1*nx)&&(idx<2*nx)){
				if(Ekall(Result[idx-nx])>=0.0)
					Result[idx] = Px(double(Result[idx-nx]));
				else Result[idx] = 0.0;
			}
			//第三列为第一列各自的fx初值，出现小于零情况同理。
			if((idx>=2*nx)&&(idx<3*nx)){
				if(Result[idx-1*nx]>0.0)
					Result[idx] = fx(double(Result[idx-2*nx]));
				else Result[idx] = 0.0;
			}
			
			//第四五六列为前三列的复制，为了compute函数准备
			if((idx>=3*nx)&&(idx<4*nx)){
				if(Result[idx-2*nx]>0.0)
					Result[idx] = Result[idx-3*nx];
				else Result[idx] = 0.0;
			}
			if((idx>=4*nx)&&(idx<5*nx))
				Result[idx] = Result[idx-3*nx];
			if((idx>=5*nx)&&(idx<6*nx))
				Result[idx] = Result[idx-3*nx];
		}
	}

}


void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //生成随机数变量
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
    hiprandGenerateNormalDouble(gen, ip, size, 0, 1);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
    hiprandDestroyGenerator(gen);                         //释放指针
	return;
	
	
}


void  InitialMatrix(double* d_Result,int nx,int ny){
	NormalRandom(d_Result,nx);
	//分配grid,block大小
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	InitialKernel<<<grid,block>>>(d_Result,nx,ny);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	
	
	
	//保存数据仅仅为了测试用，写好compute部分以后肯定不用保存这个数据了……
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	double *h_gpuRef;
	h_gpuRef = (double *)malloc(nBytes);
	CHECK(hipMemcpy(h_gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"init.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	
	
}






__device__ double updateXi(double xi,double pxi)
{
	double K1=pxi;
	double K2=xi+K1/2.0*DX;
	double K3=xi+K2/2.0*DX;
	double K4=xi+K3*DX;
	
	return xi+DX*(K1+2*K2+2*K3+K4)/6.0;
}


__device__ double updatePxi(double pxi,double fxi)
{
	double K1=fxi;
	double K2=pxi+K1/2.0*DX;
	double K3=pxi+K2/2.0*DX;
	double K4=pxi+K3*DX;
	
	return pxi+DX*(K1+2*K2+2*K3+K4)/6.0;
}




//Runge_Kutta 方法，待完成-----1118wzl;
//Runge_Kutta 方法，待完成-----1118wzl;
//Runge_Kutta 方法，待完成-----1118wzl;
//Runge_Kutta 方法，待完成-----1118wzl;
//Runge_Kutta 方法，待完成-----1118wzl;


__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx ){
		
		return;
		
		}
}







 void ComputeOnGPU1(double* Result,int nx,int ny,double* h_gpuRef){
	
	
		//分配grid,block大小
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	 CHECK(hipDeviceSynchronize());
	    //如果核函数错误，返回信息
    CHECK(hipGetLastError());
	 // GPU数据拷贝回主机
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	CHECK(hipMemcpy(h_gpuRef, Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"gpu.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	return;
}









