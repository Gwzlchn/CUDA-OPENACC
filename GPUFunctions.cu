#include "hip/hip_runtime.h"
#include"GPUFunctions.h"
#include ""
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<math.h>


__device__ double fx(double x)
{

	return -1.0/(pow(sqrt(pow(x,2.0)+pow(A,2.0)),3.0));
}


__device__ double Ekall(double x)
{

	return E0+1.0/(sqrt(pow(x,2.0)+pow(A,2.0)));
}

__device__ double Px(double x)
{
	return sqrt(2*Ekall(x));
}






void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //生成随机数变量
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
    hiprandGenerateNormalDouble(gen, ip, size, 0, 2);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
    hiprandDestroyGenerator(gen);                         //释放指针
	return;
	
	
}






__device__ double updateXi(double xi,double dx)
{
	double tempPx=Px(xi);
	double K1=tempPx,
		K2=xi+(tempPx/2.0)*dx,
		K3=xi+(((tempPx/2.0)*dx)+xi)/2.0*dx,
		K4=xi+(((((tempPx/2.0)*dx)+xi)/2.0*dx)+xi)*dx;
	
	return xi+dx*(K1+2*K2+2*K3+K4)/6.0;
}


__device__ double updatePxi(double xi,double dx)
{
	double tempPx=Px(xi),tempFx=fx(xi);
	double K1=tempFx,
		K2=tempPx+(tempFx/2.0)*dx,
		K3=tempPx+(((tempFx/2.0)*dx)+tempPx)/2.0*dx,
		K4=tempPx+(((((tempFx/2.0)*dx)+xi)/2.0*dx)+tempPx)*dx;
	
	return tempPx+dx*(K1+2*K2+2*K3+K4)/6.0;
}





__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx ){
		
            //int idx = iy * nx + ix;
             //Result[idx] = MatA[idx];
			 //Result[1*nx] = MatA[nx];
        
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
            
			if((idx>=1*nx)&&(idx<2*nx))
				Result[idx] = Px(double(Result[idx-nx]));
			if((idx>=2*nx)&&(idx<3*nx))
				Result[idx] = fx(double(Result[idx-2*nx]));
			if((idx>=3*nx)&&(idx<4*nx)){
				Result[idx] = Result[idx-3*nx];
				int i,n=1+(TOSTOP)/DX;
				for(i=1;i<n;i++)
					Result[idx]=updateXi(Result[idx],DX);
			}
			if((idx>=4*nx)&&(idx<5*nx)){
				Result[idx] = Result[idx-3*nx];
				int i,n=1+(TOSTOP)/DX;
				for(i=1;i<n;i++)
					Result[idx]=updatePxi(Result[idx],DX);
			}
			
				
				/*const double dx=0.00001;
				int i,n=1+(2*PI)/dx;
				double temp;
				for (i = 1; i < n; i++){
					temp=rk4(dx, Result[idx-2*nx] + dx * (i - 1), Result[idx]);
					Result[idx] = temp;*/
					
		}
	}

}






 void ComputeOnGPU1(double* Result,int nx,int ny,dim3 grid,dim3 block){
	
	
	
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	return;
}









