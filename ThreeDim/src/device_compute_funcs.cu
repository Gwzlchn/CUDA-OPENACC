#include "hip/hip_runtime.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <hip/hip_vector_types.h>
#include "../include/sci_const.h"
#include "../include/device_compute_funcs.h"

__device__ double E_kall(const nucleus& first, const nucleus& second)
{
	return E_total - (-1.0 / sqrt(pow((first.z - nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
		pow((first.x - nuclear_spacing - 2.0*sin(PI*rotation)), 2) +
		first.y*first.y + elec_elec*elec_elec)) -
		(-1.0 / sqrt(pow((second.z - nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
			pow((second.x - nuclear_spacing / 2.0*sin(PI*rotation)), 2) +
			second.y*second.y + elec_elec*elec_elec))
		- (1.0 / sqrt(nucleus_distance(first, second) + elec_nucl*elec_nucl)) -
		(-1.0 / sqrt(pow((first.z + nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
			pow((first.x + nuclear_spacing / 2.0*sin(PI*rotation)), 2) +
			first.y*first.y + elec_elec*elec_elec)) -
			(-1.0 / sqrt(pow((second.z + nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
				pow((second.x + nuclear_spacing / 2.0*sin(PI*rotation)), 2) +
				second.y*second.y + elec_elec*elec_elec));
}

__device__ void px_py_pz_distribution(nucleus& first, nucleus& second)
{
	double ekall = E_kall(first, second);
	hiprandStatePhilox4_32_10_t s;
	unsigned long long seed = 1;
	// seed a random number generator 
	hiprand_init(seed, 0, 0, &s);
	double2 random12 = hiprand_uniform2_double(&s);
	double2 random34 = hiprand_uniform2_double(&s);
	double random5 = hiprand_uniform_double(&s);

	double theta1 = random12.x*PI;
	double theta2 = random12.y*PI;
	double phi1 = random34.x * 2 * PI;
	double phi2 = random34.y * 2 * PI;

	first.px = sqrt(2.0*ekall*random5)*sin(theta1)*cos(phi1);
	first.py = sqrt(2.0*ekall*random5)*sin(theta1)*cos(phi1);
	first.pz = sqrt(2.0*ekall*random5)*cos(phi1);

	second.px = sqrt(2.0*ekall*(1 - random5))*sin(theta2)*cos(phi2);
	second.py = sqrt(2.0*ekall*(1 - random5))*sin(theta2)*cos(phi2);
	second.pz = sqrt(2.0*ekall*(1 - random5))*cos(phi2);



}


__device__  double nucleus_distance(const nucleus& first, const nucleus& second)
{
	return (pow((first.x - second.x), 2) + pow((first.y - second.y), 2) + pow((first.z - second.z), 2));
}




__device__  nucleus fx_first_nucleus(const nucleus& first, const nucleus& second)
{
	nucleus fx_first;

	fx_first.x = (first.x - second.x)
		/ sqrt(pow((nucleus_distance(first, second) + elec_nucl*elec_nucl), 3))
		- (first.x - nuclear_spacing / 2.0 * sin(PI*rotation))
		/ sqrt(pow((pow((first.z - nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((first.x - nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			first.y*first.y + elec_elec*elec_elec), 3))
		- (first.x + nuclear_spacing / 2.0 * sin(PI*rotation))
		/ sqrt(pow((pow((first.z + nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((first.x + nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			first.y*first.y + elec_elec*elec_elec), 3));

	fx_first.y = (first.y - second.y)
		/ sqrt(pow((nucleus_distance(first, second) + elec_nucl*elec_nucl), 3))
		- first.y
		/ sqrt(pow((pow((first.z - nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
			pow((first.x - nuclear_spacing / 2.0*sin(PI*rotation)), 2) +
			first.y*first.y + elec_elec*elec_elec), 3))
		- first.y
		/ sqrt(pow((pow((first.z + nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
			pow((first.x + nuclear_spacing / 2.0*sin(PI*rotation)), 3) +
			first.y*first.y + elec_elec*elec_elec), 3));

	fx_first.z = (first.z - second.z)
		/ sqrt(pow((nucleus_distance(first, second) + elec_nucl*elec_nucl), 3))
		- (first.z - nuclear_spacing / 2.0 * cos(PI*rotation))
		/ sqrt(pow((pow((first.z - nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((first.x - nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			first.y*first.y + elec_elec*elec_elec), 3))
		- (first.z + nuclear_spacing / 2.0 * cos(PI*rotation))
		/ sqrt(pow((pow((first.z + nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((first.x + nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			first.y*first.y + elec_elec*elec_elec), 3));

	fx_first.px = fx_first.py = fx_first.pz = 0;
	return fx_first;
}
__device__  nucleus fx_second_nucleus(const nucleus& first, const nucleus& second)
{
	nucleus fx_second;
	fx_second.x = (second.x - first.x)
		/ sqrt(pow((nucleus_distance(first, second) + elec_nucl*elec_nucl), 3))
		- (second.x - nuclear_spacing / 2.0 * sin(PI*rotation))
		/ sqrt(pow((pow((second.z - nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((second.x - nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			second.y*second.y + elec_elec*elec_elec), 3))
		- (second.x + nuclear_spacing / 2.0 * sin(PI*rotation))
		/ sqrt(pow((pow((second.z + nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((second.x + nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			second.y*second.y + elec_elec*elec_elec), 3));

	fx_second.y = (second.y - first.y)
		/ sqrt(pow((nucleus_distance(first, second) + elec_nucl*elec_nucl), 3))
		- second.y
		/ sqrt(pow((pow((second.z - nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
			pow((second.x - nuclear_spacing / 2.0*sin(PI*rotation)), 2) +
			second.y*second.y + elec_elec*elec_elec), 3))
		- second.y
		/ sqrt(pow((pow((second.z + nuclear_spacing / 2.0*cos(PI*rotation)), 2) +
			pow((second.x + nuclear_spacing / 2.0*sin(PI*rotation)), 3) +
			second.y*second.y + elec_elec*elec_elec), 3));

	fx_second.z = (first.z - second.z)
		/ sqrt(pow((nucleus_distance(first, second) + elec_nucl*elec_nucl), 3))
		- (second.z - nuclear_spacing / 2.0 * cos(PI*rotation))
		/ sqrt(pow((pow((second.z - nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((second.x - nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			second.y*second.y + elec_elec*elec_elec), 3))
		- (second.z + nuclear_spacing / 2.0 * cos(PI*rotation))
		/ sqrt(pow((pow((second.z + nuclear_spacing / 2.0 * cos(PI*rotation)), 2) +
			pow((second.x + nuclear_spacing / 2.0 * sin(PI*rotation)), 2) +
			second.y*second.y + elec_elec*elec_elec), 3));
	fx_second.px = fx_second.py = fx_second.pz = 0;
	return fx_second;
}


void update_step_one(nucleus& step_one_fir, nucleus& step_one_sec)
{


}
//__device__ void update_step_two(nucleus* step_two_fir, nucleus* step_two_sec)

//����˫����˫��̬�ֲ������
__global__ void DoubleNormalRandomArrayD(nuclei* Array, const long Size)
{
	double A1, A2, A3, A4, Ekall;
	int i = threadIdx.x;
	double temp1 = 1;
	double temp2 = 1;
	hiprandState s;

	Ekall = -1;

	while (Ekall < 0)
	{
		A2 = A4 = 2;

		while (A2 > temp1 && A4 > temp2)
		{
			A1 = hiprand_uniform_double(&s);
			A2 = hiprand_uniform_double(&s);
			A3 = hiprand_uniform_double(&s);
			A4 = hiprand_uniform_double(&s);

			A1 = (A1 - 0.5) * 20;
			A3 = (A3 - 0.5) * 20;

			temp1 = exp((-pow((A1 - nuclear_spacing / 2.0), nuclear_spacing / 2.0)) /
				(nuclear_spacing / 2.0 * pow(stddev, nuclear_spacing / 2.0)))
				+ exp((-pow((A1 + nuclear_spacing / 2.0), nuclear_spacing / 2.0)) /
				(nuclear_spacing / 2.0 * pow(stddev, nuclear_spacing / 2.0)));
			temp2 = exp((-pow((A3 - nuclear_spacing / 2.0), nuclear_spacing / 2.0)) /
				(nuclear_spacing / 2.0 * pow(stddev, nuclear_spacing / 2.0)))
				+ exp((-pow((A3 + nuclear_spacing / 2.0), nuclear_spacing / 2.0)) /
				(nuclear_spacing / 2.0 * pow(stddev, nuclear_spacing / 2.0)));
		}

		Array[i].first.x = A1 * sin(rotation*PI);
		Array[i].first.y = 0;
		Array[i].first.z = A1 * cos(rotation*PI);
		Array[i].second.x = A3 * sin(rotation*PI);
		Array[i].second.y = 0;
		Array[i].second.z = A3 * cos(rotation*PI);
		Ekall = E_kall(Array[i].first, Array[i].second);
	}
	return;
}

//����˫�����ӵ��������
extern "C" void NucleiRandomD(nuclei* Array, const long Size)
{
	int threadsPerBlock = 256;
	int threadsPerGrid = (2 * Size + threadsPerBlock - 1) / threadsPerBlock;
	DoubleNormalRandomArrayD << <threadsPerGrid, threadsPerBlock >> > (Array, Size);
}