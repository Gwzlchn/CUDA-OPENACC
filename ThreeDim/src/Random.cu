#include "hip/hip_runtime.h"
﻿#pragma comment(lib, "cudart.lib")
#pragma comment(lib, "hiprand.lib")
#include "../include/global_funcs.h"
#include "../include/sci_const.h"
#include "../include/device_compute_funcs.cuh"
#include "../include/common.hpp"
#include "../include/PrintStruct.h"

#include ""
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

//生成双精度双正态分布随机数
__global__ void DoubleNormalRandomArrayD(nuclei* Array, const long Size)
{
	double A1, A2, A3, A4;
	double Ekall = -1;
	double temp1 = 1;
	double temp2 = 1;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	hiprandState s;
	int seed = -i;
	hiprand_init(seed, 0, 0, &s);
	
	while (Ekall < 0)
	{
		A2 = A4 = 2;

		while (A2 > temp1 && A4 > temp2)
		{
			A1 = hiprand_uniform_double(&s);
			A2 = hiprand_uniform_double(&s);
			A3 = hiprand_uniform_double(&s);
			A4 = hiprand_uniform_double(&s);
			
			A1 = (A1 - 0.5) * 20;
			A3 = (A3 - 0.5) * 20;

			temp1 = exp((-pow((A1 - mean), 2)) / (mean * stddev * stddev))
				+ exp((-pow((A1 + mean), 2)) / (mean * stddev * stddev));
			temp2 = exp((-pow((A3 - mean), 2)) / (mean * stddev * stddev))
				+ exp((-pow((A3 + mean), 2)) / (mean * stddev * stddev));
		}
		//printf("%lf\t%lf\n", A1,A3);
	
		Array[i].first.x = A1 * sin(rotation*PI);
		Array[i].first.y = 0;
		Array[i].first.z = A1 * cos(rotation*PI);

		Array[i].second.x = A3 * sin(rotation*PI);
		Array[i].second.y = 0;
		Array[i].second.z = A3 * cos(rotation*PI);

		Ekall = E_kall(Array[i].first, Array[i].second);
		
		//printf("%lf\n", Ekall);
	}
	px_py_pz_distribution(Array[i].first, Array[i].second,Ekall,i);
	return;
}

//用于双核粒子的随机数化
void NucleiRandomD(nuclei* Array, const long Size)
{
	int dimx = 512;
	dim3 block(dimx);
	dim3 grid((Size + block.x - 1) / block.x, 1);
	DoubleNormalRandomArrayD <<< grid, block >>> (Array, Size);
}

void compute_on_gpu_one(const long pairs)
{
	long long nBytes = pairs * sizeof(nuclei);
	printf("Use %lld Bytes %lfMB\n", nBytes, nBytes / double(1024 * 1024));

	nuclei* test;
	nuclei* host;
	double Start = seconds();
	hipMalloc((void **)(&test), nBytes);
	host = (nuclei*)malloc(nBytes);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	NucleiRandomD(test, pairs);

	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	float costtime;
	hipEventElapsedTime(&costtime, start, stop);

	hipMemcpy(host, test, nBytes, hipMemcpyDeviceToHost);
	PrintStruct(host, pairs, "testOne.dat", costtime,0);
}

//生成双精度01均匀分布随机数
//参数:	Array:双精度数组	Size:数组长度
//void UniformRandomArrayD(double* Array, const long Size)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateUniformDouble(gen, Array, Size);					//生成0-1均匀分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}
//
////生成双精度正态分布随机数
////参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Stddev:方差(0.7)
//void NormalRandomArrayD(double* Array, const long Size, double Mean, double Stddev)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Stddev);		//生成正态分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}

////生成双精度双正态分布随机数
////参数:	Array1:双精度数组1	Array2:双精度数组2	Array3:双精度数组3	Array2:双精度数组4	
////Size:数组长度	Nudis:半核间距(2)	Stddev:方差(0.7)
//__global__ void DoubleNormalRandomArrayD(double* Array1, double* Array2, double* Array3, double* Array4,
//	const long Size )
//{
//	int i = threadIdx.x;
//	double temp1 = 1;
//	double temp2 = 1;
//
//	Array1[i] = (Array1[i] - 0.5) * 20;
//	Array3[i] = (Array3[i] - 0.5) * 20;
//
//	temp1 = exp((-pow((Array1[i] - nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)))
//		+ exp((-pow((Array1[i] + nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)));
//	temp2 = exp((-pow((Array3[i] - nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)))
//		+ exp((-pow((Array3[i] + nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)));
//
//	if (Array2[i] > temp1 && Array4[i] > temp2)
//	{
//		Array1[i] = -99;
//		Array3[i] = -99;
//	}
//	return;
//}
//
////线性传参
//__global__ void LinearTransmissionD(nuclei* Array, double* DTempArr1, double* DTempArr3, const long Size, int& i, int& j)
//{
//	int p, q;
//	hipMalloc((void **)(&p), 4);
//	hipMalloc((void **)(&q), 4);
//	hipMemcpy(&p, &i, 4, hipMemcpyHostToDevice);
//	hipMemcpy(&p, &i, 4, hipMemcpyHostToDevice);
//	while (i < Size && (i + j) < 2 * Size)
//	{
//		if (DTempArr1[i + j] == -99)
//		{
//			j++;
//		}
//		else {
//			Array[i].first.x = DTempArr1[i + j] * sin(rotation*PI);
//			Array[i].first.y = 0;
//			Array[i].first.z = DTempArr1[i + j] * cos(rotation*PI);
//			Array[i].second.x = DTempArr3[i + j] * sin(rotation*PI);
//			Array[i].second.y = 0;
//			Array[i].second.z = DTempArr3[i + j] * cos(rotation*PI);
//			i++;
//		}
//	}
//	hipMemcpy(&i, &p, 4, hipMemcpyDeviceToHost);
//	hipMemcpy(&j, &q, 4, hipMemcpyDeviceToHost);
//	return;
//}
//
////用于双核粒子的随机数化
////参数:	Array:粒子数组	Size:数组长度 Angle:偏移角
//void NucleiRandomD(nuclei* Array, const long Size)
//{
//	int i(0);
//	int j(0);
//	size_t DoubleSize = 2 * Size * sizeof(double);
//	double *DTempArr1, *DTempArr2, *DTempArr3, *DTempArr4;
//	hipMalloc((void**)&DTempArr1, DoubleSize);
//	hipMalloc((void**)&DTempArr2, DoubleSize);
//	hipMalloc((void**)&DTempArr3, DoubleSize);
//	hipMalloc((void**)&DTempArr4, DoubleSize);
//
//	while (i < Size)
//	{
//		UniformRandomArrayD(DTempArr1, 2 * Size);
//		UniformRandomArrayD(DTempArr2, 2 * Size);
//		UniformRandomArrayD(DTempArr3, 2 * Size);
//		UniformRandomArrayD(DTempArr4, 2 * Size);
//
//		int threadsPerBlock = 256;
//		int threadsPerGrid = (2 * Size + threadsPerBlock - 1) / threadsPerBlock;
//		DoubleNormalRandomArrayD <<<threadsPerGrid, threadsPerBlock >>> (DTempArr1, DTempArr2, DTempArr3, DTempArr4, 2 * Size);
//		LinearTransmissionD <<<1,1>>>(Array, DTempArr1, DTempArr3, Size, i, j);
//	}
//}