﻿#include "../include/global_funcs.h"
#include <hiprand.h>
#include ""

//生成双精度01均匀分布随机数
//参数:	Array:双精度数组	Size:数组长度
void UniformRandomArray(double* Array, const long Size)
{
	hiprandGenerator_t gen;											//生成随机数变量
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
	hiprandGenerateUniformDouble(gen, Array, Size);					//生成0-1均匀分布随机数，存储到缓冲器中
	hiprandDestroyGenerator(gen);                         			//释放指针
	return;
}

//生成双精度正态分布随机数
//参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Stddev:方差(0.7)
void NormalRandomArray(double* Array, const long Size, double Mean , double Stddev)
{
	hiprandGenerator_t gen;											//生成随机数变量
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Stddev);		//生成正态分布随机数，存储到缓冲器中
	hiprandDestroyGenerator(gen);                         			//释放指针
	return;
}

//生成双精度双正态分布随机数
//参数:	Array:双精度数组	Size:数组长度	Nudis:核间距(2)	Stddev:方差(0.7)
void DoubleNormalRandomArray(double* Array, const long Size, double Nudis, double Stddev)
{
	UniformRandomArray(Array, Size);
	return;
}

//用于粒子的随机数化
//参数:	Array:粒子数组	Size:数组长度
void NucleusRandom(nucleus* Array, const long Size)
{
	double* arr = new double[Size];
	NormalRandomArray(arr, Size);
	for (int i = 0; i < Size; i++)
	{

	}
}

void NormalRandomNuclei(nuclei* raw_nuclei,double* random_arr ,const long n)
{
	for(long i=0;i<(n/2);i++)
	{
		raw_nuclei[i].init_first.x = random_arr[i];
		raw_nuclei[i].init_second.x = random_arr[i + (n / 2)];
	}
}

void NormalRandom(nuclei* raw_nuclei, const long n)
{

}
void InitialNuclei(nuclei* randomed_nuclei, const long raw_count, long & left)
{

}
void FirstStep(nuclei* inited_nuclei, const long n)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

}