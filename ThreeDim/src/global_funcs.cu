﻿#include "../include/global_funcs.h"
#include<hiprand.h>

void NormalRandomArray(double* arr, const long n2)
{
	hiprandGenerator_t gen;                                  //生成随机数变量
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
	hiprandGenerateNormalDouble(gen, arr, n2, 0, 0.7);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
	hiprandDestroyGenerator(gen);                         	//释放指针
	return;
}
void NormalRandomNuclei(nuclei* raw_nuclei,double* random_arr ,const long n)
{
	for(long i=0;i<(n/2);i++)
	{
		raw_nuclei[i].init_first.x = random_arr[i];
		raw_nuclei[i].init_second.x = random_arr[i + (n / 2)];
	}
}