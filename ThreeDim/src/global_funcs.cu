#include "hip/hip_runtime.h"
﻿#pragma comment(lib, "cudart.lib")
#pragma comment(lib, "hiprand.lib")
#include "../include/global_funcs.h"
#include "../include/sci_const.h"
#include "../include/device_compute_funcs.cuh"
#include "../include/common.hpp"
#include "../include/PrintStruct.h"

#include ""
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

//生成双精度01均匀分布随机数
//参数:	Array:双精度数组	Size:数组长度
//void UniformRandomArrayD(double* Array, const long Size)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateUniformDouble(gen, Array, Size);					//生成0-1均匀分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}
//
////生成双精度正态分布随机数
////参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Stddev:方差(0.7)
//void NormalRandomArrayD(double* Array, const long Size, double Mean, double Stddev)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Stddev);		//生成正态分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}

////生成双精度双正态分布随机数
////参数:	Array1:双精度数组1	Array2:双精度数组2	Array3:双精度数组3	Array2:双精度数组4	
////Size:数组长度	Nudis:半核间距(2)	Stddev:方差(0.7)
//__global__ void DoubleNormalRandomArrayD(double* Array1, double* Array2, double* Array3, double* Array4,
//	const long Size )
//{
//	int i = threadIdx.x;
//	double temp1 = 1;
//	double temp2 = 1;
//
//	Array1[i] = (Array1[i] - 0.5) * 20;
//	Array3[i] = (Array3[i] - 0.5) * 20;
//
//	temp1 = exp((-pow((Array1[i] - nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)))
//		+ exp((-pow((Array1[i] + nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)));
//	temp2 = exp((-pow((Array3[i] - nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)))
//		+ exp((-pow((Array3[i] + nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)));
//
//	if (Array2[i] > temp1 && Array4[i] > temp2)
//	{
//		Array1[i] = -99;
//		Array3[i] = -99;
//	}
//	return;
//}
//
////线性传参
//__global__ void LinearTransmissionD(nuclei* Array, double* DTempArr1, double* DTempArr3, const long Size, int& i, int& j)
//{
//	int p, q;
//	hipMalloc((void **)(&p), 4);
//	hipMalloc((void **)(&q), 4);
//	hipMemcpy(&p, &i, 4, hipMemcpyHostToDevice);
//	hipMemcpy(&p, &i, 4, hipMemcpyHostToDevice);
//	while (i < Size && (i + j) < 2 * Size)
//	{
//		if (DTempArr1[i + j] == -99)
//		{
//			j++;
//		}
//		else {
//			Array[i].first.x = DTempArr1[i + j] * sin(rotation*PI);
//			Array[i].first.y = 0;
//			Array[i].first.z = DTempArr1[i + j] * cos(rotation*PI);
//			Array[i].second.x = DTempArr3[i + j] * sin(rotation*PI);
//			Array[i].second.y = 0;
//			Array[i].second.z = DTempArr3[i + j] * cos(rotation*PI);
//			i++;
//		}
//	}
//	hipMemcpy(&i, &p, 4, hipMemcpyDeviceToHost);
//	hipMemcpy(&j, &q, 4, hipMemcpyDeviceToHost);
//	return;
//}
//
////用于双核粒子的随机数化
////参数:	Array:粒子数组	Size:数组长度 Angle:偏移角
//void NucleiRandomD(nuclei* Array, const long Size)
//{
//	int i(0);
//	int j(0);
//	size_t DoubleSize = 2 * Size * sizeof(double);
//	double *DTempArr1, *DTempArr2, *DTempArr3, *DTempArr4;
//	hipMalloc((void**)&DTempArr1, DoubleSize);
//	hipMalloc((void**)&DTempArr2, DoubleSize);
//	hipMalloc((void**)&DTempArr3, DoubleSize);
//	hipMalloc((void**)&DTempArr4, DoubleSize);
//
//	while (i < Size)
//	{
//		UniformRandomArrayD(DTempArr1, 2 * Size);
//		UniformRandomArrayD(DTempArr2, 2 * Size);
//		UniformRandomArrayD(DTempArr3, 2 * Size);
//		UniformRandomArrayD(DTempArr4, 2 * Size);
//
//		int threadsPerBlock = 256;
//		int threadsPerGrid = (2 * Size + threadsPerBlock - 1) / threadsPerBlock;
//		DoubleNormalRandomArrayD <<<threadsPerGrid, threadsPerBlock >>> (DTempArr1, DTempArr2, DTempArr3, DTempArr4, 2 * Size);
//		LinearTransmissionD <<<1,1>>>(Array, DTempArr1, DTempArr3, Size, i, j);
//	}
//}

//生成双精度双正态分布随机数double3

__global__ void DoubleNormalRandomArrayD(nuclei* Array, const long Size)
{
	
	double A1, A2, A3, A4;
	double Ekall = -1;
	double temp1 = 1;
	double temp2 = 1;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < Size)
	{
		hiprandState s;
		int seed = -i;
		hiprand_init(seed, 0, 0, &s);

		while (Ekall < 0)
		{
			A2 = A4 = 2;

			while (A2 > temp1 && A4 > temp2)
			{
				A1 = hiprand_uniform_double(&s);
				A2 = hiprand_uniform_double(&s);
				A3 = hiprand_uniform_double(&s);
				A4 = hiprand_uniform_double(&s);

				A1 = (A1 - 0.5) * 20;
				A3 = (A3 - 0.5) * 20;

				temp1 = exp((-pow((A1 - mean), 2)) / (mean * stddev * stddev))
					+ exp((-pow((A1 + mean), 2)) / (mean * stddev * stddev));
				temp2 = exp((-pow((A3 - mean), 2)) / (mean * stddev * stddev))
					+ exp((-pow((A3 + mean), 2)) / (mean * stddev * stddev));
			}
			//printf("%lf\t%lf\n", A1,A3);

			Array[i].first.x = A1 * sin(rotation*PI);
			Array[i].first.y = 0;
			Array[i].first.z = A1 * cos(rotation*PI);

			Array[i].second.x = A3 * sin(rotation*PI);
			Array[i].second.y = 0;
			Array[i].second.z = A3 * cos(rotation*PI);

			Ekall = E_kall(Array[i].first, Array[i].second);

			//printf("%lf\n", Ekall);
		}
		px_py_pz_distribution(Array[i].first, Array[i].second, Ekall, i);
	}
	return;
}

__global__ void first_step_on_gpu(nuclei* first_arr, const long size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("%p\n", &first_arr);
	if(idx<size)
	{
		//printf("%d\n", idx);
		for (int i = 0; i < one_steps; i++)
			update_step_one(first_arr[idx].first, first_arr[idx].second);
	}
	
	
}


__global__ void pre_second_step(double* AW)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	double t0 = 2 * PI / omega;
	double t1 = 0.5 * DX * idx; 
	AW[idx] = field_strength / omega * pow(sin(PI * t1) / (10 * t0), 2) * cos(omega * t1);
}

__global__



__global__ void second_step_on_gpu(nuclei* second_arr, const long size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx<size)
	{
		for (int i = 0; i < two_steps; i++)
			update_step_two(second_arr[idx].first, second_arr[idx].second);
	}
}

//用于双核粒子的随机数化
void NucleiRandomD(nuclei* Array, const long Size)
{
	int dimx = 512;
	dim3 block(dimx);
	dim3 grid((Size + block.x - 1) / block.x, 1);
	DoubleNormalRandomArrayD <<< grid, block >>> (Array, Size);
	//hipDeviceSynchronize();
}


void NucleiFisrtStep(nuclei* first_array, const long size)
{
	int dimx = 8;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	first_step_on_gpu <<< grid, block >>> (first_array, size);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}
	//hipDeviceSynchronize();
	printf("123\n");
}




void NucleiSecondStep(nuclei* second_array, const long size)
{
	int dimx = 512;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	second_step_on_gpu <<< grid, block >>> (second_array, size);
	//hipDeviceSynchronize();
}



void compute_on_gpu_one(const long pairs,const char* file_name)
{
	long long nBytes = pairs * sizeof(nuclei);
	printf("Use %lld Bytes %lfMB\n", nBytes, nBytes / double(1024 * 1024));
	nuclei *gpu_init,*gpu_first,*gpu_second;
	nuclei *host_init,*host_first,*host_second;
	host_init = (nuclei*)malloc(nBytes);
	host_first = (nuclei*)malloc(nBytes);
	host_second = (nuclei*)malloc(nBytes);



	//初始化！
	//申请init空间
	double start = seconds();
	CHECK(hipMalloc((void **)(&gpu_init), nBytes));
	//计算
	NucleiRandomD(gpu_init, pairs);

	//把值赋给第一步(也申请了第一步的空间)
	CHECK(hipMalloc((void **)(&gpu_first), nBytes));
	CHECK(hipMemcpy(gpu_first, gpu_init, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_init, gpu_init, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipDeviceSynchronize());
	PrintStruct(host_init, pairs, file_name, 0);
	//释放init空间
	//CHECK(hipFree(gpu_init));
	double elapse = seconds();
	printf("Inition compltete %lf\n", elapse - start);
	//初始化完成！


	//第一步计算
	//first空间在之前申请过了
	 start = seconds();
	//计算
	NucleiFisrtStep(gpu_first, pairs);
	CHECK(hipDeviceSynchronize());

	//把值赋给第二步(也申请了第二步的空间)
	/*CHECK(hipMalloc((void **)(&gpu_second), nBytes));
	CHECK(hipMemcpy(gpu_second, gpu_first, nBytes, hipMemcpyDeviceToDevice));*/
	//拷回并保存
	CHECK(hipMemcpy(host_first, gpu_first, nBytes, hipMemcpyDeviceToHost));
	PrintStruct(host_first, pairs, file_name, 1);
	//释放first空间
	//CHECK(hipFree(gpu_first));
	 elapse = seconds();
	printf("FirstStep compltete %lf\n", elapse - start);
	//第一步完成！


	////第二步计算
	//start = seconds();
	////计算
	//NucleiSecondStep(gpu_second, pairs);

	////拷回并保存
	//CHECK(hipMemcpy(host_second, gpu_second, nBytes, hipMemcpyDeviceToHost));
	//
	//PrintStruct(host_second, pairs,file_name , 2);
	////释放second空间
	//CHECK(hipFree(gpu_second));
	//
	//elapse = seconds();
	//printf("SecondStep compltete %lf\n", elapse - start);
	//// 第二步完成！
	//


	return;
}