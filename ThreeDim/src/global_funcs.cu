#include "hip/hip_runtime.h"
﻿#pragma comment(lib, "cudart.lib")
#pragma comment(lib, "hiprand.lib")
#include "../include/global_funcs.h"
#include "../include/sci_const.h"
#include "../include/device_compute_funcs.cuh"
#include "../include/common.hpp"
#include "../include/PrintStruct.h"

#include ""
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

//生成双精度01均匀分布随机数
//参数:	Array:双精度数组	Size:数组长度
//void UniformRandomArrayD(double* Array, const long Size)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateUniformDouble(gen, Array, Size);					//生成0-1均匀分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}
//
////生成双精度正态分布随机数
////参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Stddev:方差(0.7)
//void NormalRandomArrayD(double* Array, const long Size, double Mean, double Stddev)
//{
//	hiprandGenerator_t gen;											//生成随机数变量
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//指定算法
//	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//随机数初始化
//	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Stddev);		//生成正态分布随机数，存储到缓冲器中
//	hiprandDestroyGenerator(gen);                         			//释放指针
//	return;
//}

////生成双精度双正态分布随机数
////参数:	Array1:双精度数组1	Array2:双精度数组2	Array3:双精度数组3	Array2:双精度数组4	
////Size:数组长度	Nudis:半核间距(2)	Stddev:方差(0.7)
//__global__ void DoubleNormalRandomArrayD(double* Array1, double* Array2, double* Array3, double* Array4,
//	const long Size )
//{
//	int i = threadIdx.x;
//	double temp1 = 1;
//	double temp2 = 1;
//
//	Array1[i] = (Array1[i] - 0.5) * 20;
//	Array3[i] = (Array3[i] - 0.5) * 20;
//
//	temp1 = exp((-pow((Array1[i] - nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)))
//		+ exp((-pow((Array1[i] + nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)));
//	temp2 = exp((-pow((Array3[i] - nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)))
//		+ exp((-pow((Array3[i] + nuclear_spacing/2.0), nuclear_spacing/2.0)) / (nuclear_spacing/2.0 * pow(stddev, nuclear_spacing/2.0)));
//
//	if (Array2[i] > temp1 && Array4[i] > temp2)
//	{
//		Array1[i] = -99;
//		Array3[i] = -99;
//	}
//	return;
//}
//
////线性传参
//__global__ void LinearTransmissionD(nuclei* Array, double* DTempArr1, double* DTempArr3, const long Size, int& i, int& j)
//{
//	int p, q;
//	hipMalloc((void **)(&p), 4);
//	hipMalloc((void **)(&q), 4);
//	hipMemcpy(&p, &i, 4, hipMemcpyHostToDevice);
//	hipMemcpy(&p, &i, 4, hipMemcpyHostToDevice);
//	while (i < Size && (i + j) < 2 * Size)
//	{
//		if (DTempArr1[i + j] == -99)
//		{
//			j++;
//		}
//		else {
//			Array[i].first.x = DTempArr1[i + j] * sin(rotation*PI);
//			Array[i].first.y = 0;
//			Array[i].first.z = DTempArr1[i + j] * cos(rotation*PI);
//			Array[i].second.x = DTempArr3[i + j] * sin(rotation*PI);
//			Array[i].second.y = 0;
//			Array[i].second.z = DTempArr3[i + j] * cos(rotation*PI);
//			i++;
//		}
//	}
//	hipMemcpy(&i, &p, 4, hipMemcpyDeviceToHost);
//	hipMemcpy(&j, &q, 4, hipMemcpyDeviceToHost);
//	return;
//}
//
////用于双核粒子的随机数化
////参数:	Array:粒子数组	Size:数组长度 Angle:偏移角
//void NucleiRandomD(nuclei* Array, const long Size)
//{
//	int i(0);
//	int j(0);
//	size_t DoubleSize = 2 * Size * sizeof(double);
//	double *DTempArr1, *DTempArr2, *DTempArr3, *DTempArr4;
//	hipMalloc((void**)&DTempArr1, DoubleSize);
//	hipMalloc((void**)&DTempArr2, DoubleSize);
//	hipMalloc((void**)&DTempArr3, DoubleSize);
//	hipMalloc((void**)&DTempArr4, DoubleSize);
//
//	while (i < Size)
//	{
//		UniformRandomArrayD(DTempArr1, 2 * Size);
//		UniformRandomArrayD(DTempArr2, 2 * Size);
//		UniformRandomArrayD(DTempArr3, 2 * Size);
//		UniformRandomArrayD(DTempArr4, 2 * Size);
//
//		int threadsPerBlock = 256;
//		int threadsPerGrid = (2 * Size + threadsPerBlock - 1) / threadsPerBlock;
//		DoubleNormalRandomArrayD <<<threadsPerGrid, threadsPerBlock >>> (DTempArr1, DTempArr2, DTempArr3, DTempArr4, 2 * Size);
//		LinearTransmissionD <<<1,1>>>(Array, DTempArr1, DTempArr3, Size, i, j);
//	}
//}

//生成双精度双正态分布随机数double3

__global__ void DoubleNormalRandomArrayD(nuclei* Array, const long Size)
{
	
	double A1, A2, A3, A4;
	double Ekall = -1;
	double temp1 = 1;
	double temp2 = 1;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < Size)
	{
		hiprandState s;
		int seed = -i;
		hiprand_init(seed, 0, 0, &s);

		while (Ekall < 0)
		{
			A2 = A4 = 2;

			while (A2 > temp1 && A4 > temp2)
			{
				A1 = hiprand_uniform_double(&s);
				A2 = hiprand_uniform_double(&s);
				A3 = hiprand_uniform_double(&s);
				A4 = hiprand_uniform_double(&s);

				A1 = (A1 - 0.5) * 20;
				A3 = (A3 - 0.5) * 20;

				temp1 = exp((-pow((A1 - mean), 2)) / (mean * stddev * stddev))
					+ exp((-pow((A1 + mean), 2)) / (mean * stddev * stddev));
				temp2 = exp((-pow((A3 - mean), 2)) / (mean * stddev * stddev))
					+ exp((-pow((A3 + mean), 2)) / (mean * stddev * stddev));
			}
			//printf("%lf\t%lf\n", A1,A3);

			Array[i].first.x = A1 * sin(rotation*PI);
			Array[i].first.y = 0;
			Array[i].first.z = A1 * cos(rotation*PI);

			Array[i].second.x = A3 * sin(rotation*PI);
			Array[i].second.y = 0;
			Array[i].second.z = A3 * cos(rotation*PI);

			Ekall = E_kall(Array[i].first, Array[i].second);

			//printf("%lf\n", Ekall);
		}
		px_py_pz_distribution(Array[i].first, Array[i].second, Ekall, i);
	}
	return;
}

__global__ void first_step_on_gpu(nuclei* first_arr, const long size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("%p\n", &first_arr);
	if(idx<size)
	{
		//printf("%d\n", idx);
		for (int i = 0; i < one_steps; i++)
			update_step_one(first_arr[idx].first, first_arr[idx].second);
	}
	
	
}


__global__ void pre_second_step_aw(double* AW)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2 * two_steps)
	{

		double field_strength = sqrt((2.8e15) / (3.51e16)); // 场强，对应之前ee0

		double t0 = 2 * PI / omega;
		double t1 = 0.5 * DX * idx;
		AW[idx] = (field_strength / omega) * (pow(sin((PI * t1) / (10 * t0)), 2)) * cos(omega * t1);
	}
	
}

__global__ void pre_second_step_ds(double* AW,double* DS)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < 2*two_steps)
	{
		if (idx == 0)
			DS[idx] = (AW[1] - AW[0]) / (0.5*DX);
		if (idx == (2 * two_steps - 1))
			DS[idx] = (AW[idx] - AW[idx - 1]) / (0.5*DX);
		else
		{
			DS[idx] = (AW[idx + 1] - AW[idx - 1]) / 2.0 /(0.5* DX);
		}
	}

}


__global__ void second_step_on_gpu(nuclei* second_arr, nuclei* second_arr_fliter , const long size,double* DS,unsigned long long* ee1_ee2_count)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	double e_laser_t1=0.0, e_laser_t2=0.0, e_laser_t3=0.0, e_laser_t4=0.0;
	int idx_of_ds=-1; // 相当于nn
	double t1=0.0, t2=0.0, t3=0.0, t4=0.0;
	double now_t=0.0; //当前时间，相当于t(1)
	if (idx<size)
	{
		for (int i = 0; i < two_steps; i++)
		{
			//第一个激光场强度
			t1 = now_t;
			if (t1 == 0)
				e_laser_t1 = 0.0;
			else
			{
				idx_of_ds = (2.0 * t1) / DX - 1;
				e_laser_t1 = DS[idx_of_ds];
			}
			//第二个激光场强度
			t2 = now_t + DX / 2.0;
			idx_of_ds = 2.0 * t2 / DX- 1;
			e_laser_t2 = DS[idx_of_ds];
			//第三个激光场强度
			t3 = now_t + DX / 2.0;
			idx_of_ds = 2 * t3 / DX- 1;
			e_laser_t3 = DS[idx_of_ds];
			//第四个激光场强度
			t4 = now_t + DX;
			idx_of_ds = 2.0 * t4 / DX - 1;
			e_laser_t4 = DS[idx_of_ds];
			update_step_two(second_arr[idx].first, second_arr[idx].second,
							e_laser_t1,e_laser_t2,e_laser_t3,e_laser_t4);
			now_t = now_t + DX;
			/*if(idx_of_ds == -1 )
				update_step_two(second_arr[idx].first, second_arr[idx].second,
									0.0,DS[0],DS[0],DS[1]);
			else
			{
				update_step_two(second_arr[idx].first, second_arr[idx].second,
					DS[idx_of_ds], DS[idx_of_ds + 1], DS[idx_of_ds + 1], DS[idx_of_ds + 2]);
			}
			idx_of_ds += 2;*/

		}
		double ee1 = CalculationE1(second_arr[idx].first, second_arr[idx].second);
		double ee2 = CalculationE2(second_arr[idx].first, second_arr[idx].second);
		if (ee1>0 && ee2>0)
		{
			nuclei temp;
			temp.first = second_arr[idx].first;
			temp.second = second_arr[idx].second;
			second_arr_fliter[*ee1_ee2_count] = temp;
			atomicAdd(ee1_ee2_count, 1);
		}
			
	}
}

//用于双核粒子的随机数化
void NucleiRandomD(nuclei* Array, const long Size)
{
	int dimx = 512;
	dim3 block(dimx);
	dim3 grid((Size + block.x - 1) / block.x, 1);
	DoubleNormalRandomArrayD <<< grid, block >>> (Array, Size);
	CHECK(hipDeviceSynchronize());
}


void NucleiFisrtStep(nuclei* first_array, const long size)
{
	int dimx = 32;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	first_step_on_gpu <<< grid, block >>> (first_array, size);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "1st Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}
	CHECK(hipDeviceSynchronize());
	
}




void NucleiSecondStep(nuclei* second_array, nuclei* second_array_fliter, const long size, double* aw, double* ds, unsigned long long* count)
{
	//准备矢量势
	int pre_dimx = 512;
	dim3 pre_block(pre_dimx);
	dim3 pre_grid((2 * two_steps_in_host + pre_block.x - 1) / pre_block.x, 1);
	pre_second_step_aw <<< pre_grid,pre_block>>> (aw);
	CHECK(hipDeviceSynchronize());
	pre_second_step_ds <<< pre_grid, pre_block >>> (aw, ds);
	CHECK(hipDeviceSynchronize());
	
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "2nd Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	//计算第二步
	int dimx = 32;
	dim3 block(dimx);
	dim3 grid((size + block.x - 1) / block.x, 1);
	second_step_on_gpu <<< grid, block >>> (second_array, size, ds,count);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "2nd Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}
	CHECK(hipDeviceSynchronize());
}



void compute_on_gpu_one(const long pairs,const char* file_name)
{
	long long nBytes = pairs * sizeof(nuclei);
	printf("Use %lld Bytes %lfMB\n", nBytes, nBytes / double(1024 * 1024));
	nuclei *gpu_init,*gpu_first,*gpu_second,*gpu_second_fliter;
	nuclei *host_init,*host_first,*host_second,*host_second_fliter;
	host_init = (nuclei*)malloc(nBytes);
	host_first = (nuclei*)malloc(nBytes);
	host_second = (nuclei*)malloc(nBytes);
	host_second_fliter = (nuclei*)malloc(nBytes);


	//初始化！
	//申请init空间
	double start = seconds();
	CHECK(hipMalloc((void **)(&gpu_init), nBytes));
	//计算
	NucleiRandomD(gpu_init, pairs);

	//把值赋给第一步(也申请了第一步的空间)
	CHECK(hipMalloc((void **)(&gpu_first), nBytes));
	CHECK(hipMemcpy(gpu_first, gpu_init, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_init, gpu_init, nBytes, hipMemcpyDeviceToHost));
	
	PrintStruct(host_init, pairs, file_name, 0);
	//释放init空间
	CHECK(hipFree(gpu_init));
	double elapse = seconds();
	printf("Inition compltete %lf\n", elapse - start);
	//初始化完成！


	//第一步计算
	//first空间在之前申请过了
	 start = seconds();
	//计算
	NucleiFisrtStep(gpu_first, pairs);
	

	//把值赋给第二步(也申请了第二步的空间)
	CHECK(hipMalloc((void **)(&gpu_second), nBytes));
	CHECK(hipMemcpy(gpu_second, gpu_first, nBytes, hipMemcpyDeviceToDevice));
	//拷回并保存
	CHECK(hipMemcpy(host_first, gpu_first, nBytes, hipMemcpyDeviceToHost));
	PrintStruct(host_first, pairs, file_name, 1);
	//释放first空间
	CHECK(hipFree(gpu_first));
	elapse = seconds();
	printf("FirstStep compltete %lf\n", elapse - start);
	//第一步完成！


	//第二步计算
	start = seconds();
	//准备导数
	double *gpu_aw, *gpu_ds;
	double *host_aw, *host_ds;
	long bytes_of_aw_ds = sizeof(double) * 2 * two_steps_in_host;
	CHECK(hipMalloc((void **)(&gpu_aw), bytes_of_aw_ds));
	CHECK(hipMalloc((void **)(&gpu_ds), bytes_of_aw_ds));
	host_aw = (double*)malloc(bytes_of_aw_ds);
	host_ds = (double*)malloc(bytes_of_aw_ds);

	
	//电离率计数
	unsigned long long*gpu_count,*host_count;
	int bytes_of_u_long = sizeof(unsigned long long);
	host_count = (unsigned long long*)malloc(bytes_of_u_long);
	CHECK(hipMalloc((void **)(&gpu_count), bytes_of_u_long));
	CHECK(hipMalloc((void **)(&gpu_second_fliter), nBytes));

	//计算

	NucleiSecondStep(gpu_second, gpu_second_fliter,pairs, gpu_aw, gpu_ds,gpu_count);

	//拷回并保存
	CHECK(hipMemcpy(host_second, gpu_second, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_second_fliter, gpu_second_fliter, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_aw, gpu_aw, bytes_of_aw_ds, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_ds, gpu_ds, bytes_of_aw_ds, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(host_count, gpu_count, bytes_of_u_long, hipMemcpyDeviceToHost));
	printf("%ld\n", *host_count);

	PrintStruct(host_second, pairs,file_name , 2);
	PrintArray(host_aw, 2 * two_steps_in_host, file_name, 0);
	PrintArray(host_ds, 2 * two_steps_in_host, file_name, 1);
	PrintStruct(host_second_fliter, *host_count, file_name, 3);
	//释放second空间
	CHECK(hipFree(gpu_second));
	CHECK(hipFree(gpu_aw));
	CHECK(hipFree(gpu_ds));
	
	elapse = seconds();
	printf("SecondStep compltete %lf\n", elapse - start);
	// 第二步完成！

	//释放主机内存空间
	//free(host_aw);
	//free(host_ds);
	//free(host_first);
	//free(host_init);
	//free(host_init);




	return;
}