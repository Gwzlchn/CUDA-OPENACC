﻿#include "../include/global_funcs.h"
#include <hiprand.h>

//生成双精度正态分布随机数
//参数:	Array:双精度数组	Size:数组长度	Mean:均值(0)	Variance:方差(0.7)
void NormalRandomArray(double* Array, const long Size, int Mean=0, double Variance=0.7)
{
	hiprandGenerator_t gen;											//生成随机数变量
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//步骤1：指定算法
	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//步骤2：随机数初始化
	hiprandGenerateNormalDouble(gen, Array, Size, Mean, Variance);	//步骤3：生成随机数，存储到缓冲器中
	hiprandDestroyGenerator(gen);                         			//释放指针
	return;
}

//生成双精度双正态分布随机数
//参数:	Array:双精度数组	Size:数组长度	Nudis:核间距(2)	Variance:方差(0.7)
void DoubleNormalRandomArray(double* Array, const long Size, int Nudis = 2, double Variance = 0.7)
{
	hiprandGenerator_t gen;											//生成随机数变量
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);		//步骤1：指定算法
	hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);					//步骤2：随机数初始化
	hiprandGenerateNormalDouble(gen, Array, Size, Nudis, Variance);	//步骤3：生成随机数，存储到缓冲器中
	hiprandDestroyGenerator(gen);                         			//释放指针
	return;
}

//用于粒子的随机数化
//参数:	Array:粒子数组	Size:数组长度
void NucleusRandom(nucleus* Array, const long Size)
{
	double* arr = new double[Size];
	NormalRandomArray(arr, Size);
	for (int i = 0; i < Size; i++)
	{

	}
}

void NormalRandomNuclei(nuclei* raw_nuclei,double* random_arr ,const long n)
{
	for(long i=0;i<(n/2);i++)
	{
		raw_nuclei[i].init_first.x = random_arr[i];
		raw_nuclei[i].init_second.x = random_arr[i + (n / 2)];
	}
}

void NormalRandom(nuclei* raw_nuclei, const long n)
{

}
void InitialNuclei(nuclei* randomed_nuclei, const long raw_count, long & left)
{

}
void FirstStep(nuclei* inited_nuclei, const long n)
{

}