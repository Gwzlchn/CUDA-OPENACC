#include "hip/hip_runtime.h"
#include"kernel_funcs.h"
#include"device_funcs.cuh"
#include<hiprand.h>
#include "common.hpp"
#include"host_funcs.hpp"

__global__ void InitialKernel(double* Result,int nx,int ny)
{
	//��һ���Ѿ����������
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx ){
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
			//�ڶ���Ϊ��һ�и��Ե�px��ֵ��������ָ�����С����������ֱ�Ӹ�ֵ0�����㲿���жϼ�Щ��nan�ж��ܷ�������
            if((idx>=1*nx)&&(idx<2*nx)){
				if(Ekall(Result[idx-nx])>=0.0)
					Result[idx] = Px(double(Result[idx-nx]));
				else Result[idx] = 0.0;
			}
			//������Ϊ��һ�и��Ե�fx��ֵ������С�������ͬ��
			if((idx>=2*nx)&&(idx<3*nx)){
				if(Result[idx-1*nx]>0.0)
					Result[idx] = fx(double(Result[idx-2*nx]));
				else Result[idx] = 0.0;
			}
			
			//����������Ϊǰ���еĸ��ƣ�Ϊ��compute����׼��
			if((idx>=3*nx)&&(idx<4*nx)){
				if(Result[idx-2*nx]>0.0)
					Result[idx] = Result[idx-3*nx];
				else Result[idx] = 0.0;
			}
			if((idx>=4*nx)&&(idx<5*nx))
				Result[idx] = Result[idx-3*nx];
			
		}
	}

}



void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //�������������
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//����1��ָ���㷨
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //����2���������ʼ��
    hiprandGenerateNormalDouble(gen, ip, size, 0, 0.7);        //����3��������������洢���������У���1������Ϊ��ֵ���ڶ���Ϊ���
    hiprandDestroyGenerator(gen);                         	//�ͷ�ָ��
	return;
	
	
}


void  InitialMatrix(double* d_Result,int nx,int ny){
	NormalRandom(d_Result,nx);
	//����grid,block��С
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	InitialKernel<<<grid,block>>>(d_Result,nx,ny);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	
	
	
	//�������ݽ���Ϊ�˲����ã�д��compute�����Ժ�϶����ñ�����������ˡ���
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	double *h_gpuRef;
	h_gpuRef = (double *)malloc(nBytes);
	CHECK(hipMemcpy(h_gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));
	//��������
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"init.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE InitialKernel DATA elapsed %lf sec\n",iElaps);
	
	
}


__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int idxOfXi  = 3 * nx + ix;
	unsigned int idxOfPxi = 4 * nx + ix;
	unsigned int idxOfXiTwo  = 5 * nx + ix;
	unsigned int idxOfPxiTwo = 6 * nx + ix;
	unsigned int idxOfTemp = 7 * nx + ix;


    if(ix<nx && Result[idxOfXi]!=0.0){
		for(int i=0;i<STEPSFIRST;i++){
			updateXi(Result[idxOfXi],Result[idxOfPxi]);
		}
		Result[idxOfXiTwo] = Result[idxOfXi];
		Result[idxOfPxiTwo] = Result[idxOfPxi];
		
		for(int i=0;i<STEPSSECOND;i++){
			updateXiAtStepTwo(Result[idxOfXiTwo],Result[idxOfPxiTwo],i*DX);
		}
		
		double TempE=0.5 * (pow(Result[idxOfPxiTwo],2.0)) - (1.0 / sqrt( pow(Result[idxOfXiTwo],2.0)+ pow(A,2.0)));
		if( TempE <= 0.0)
			Result[idxOfTemp]=-999;
	}
}



int CountZeros(double* h_Result,int nx)
{

	unsigned int idxOfXi  = nx ;
	unsigned int idxOfTemp = 7 * nx ;
	int count=0;
	for(int i=0;i<nx;i++){
		if(h_Result[idxOfXi+i] == 0.0f) count++;
		//if(h_Result[idxOfTemp+i] == -999) nonZeros++;
	}
	
	
	
	return count;
}

int CountTooBig(double* h_Result,int nx)
{

	unsigned int idxOfXi  = nx ;
	unsigned int idxOfTemp = 7 * nx ;
	int count=0;
	for(int i=0;i<nx;i++){
		//if(h_Result[idxOfXi+i] == 0.0) count++;
		if(h_Result[idxOfTemp+i] == -999) count++;
	}
	
	
	
	return count;
}









 void ComputeOnGPU1(double* Result,int nx,int ny,double* h_gpuRef){
	
	
	//����grid,block��С
	int dimx = 512;
    dim3 block(dimx);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	
	double iStart = seconds();
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	 CHECK(hipDeviceSynchronize());
	//����˺������󣬷�����Ϣ
    CHECK(hipGetLastError());
	double iElaps = seconds() - iStart;
	printf("RungeOnGPU  elapsed %f sec\n",iElaps);
	
	// GPU���ݿ���������
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	CHECK(hipMemcpy(h_gpuRef, Result, nBytes, hipMemcpyDeviceToHost));
	
	int zeros=0,nonzeros=0;
	zeros = CountZeros(h_gpuRef,nx);
	nonzeros = CountTooBig(h_gpuRef,nx);
	printf("The Number of Zeros is %d,\t The Number of NonZeros is %d \n",zeros,nonzeros);
	double per = (nx - zeros - nonzeros)/(nx - zeros);
	printf("Percentage is %lf  \n",per);
	
	//��������
	iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"gpuStepTwo1202.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	iElaps = seconds() - iStart;
    printf("STORE THE ComputeKernel DATA elapsed %lf sec\n",iElaps);
	return;
}

