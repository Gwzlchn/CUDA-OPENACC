#include "hip/hip_runtime.h"
#include"kernel_funcs.h"
#include"device_funcs.cuh"
#include<hiprand.h>
#include "common.hpp"
#include"host_funcs.hpp"

__global__ void InitialKernel(double* Result,int nx,int ny)
{
	//第一列已经是随机数了
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx ){
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
			//第二列为第一列各自的px初值，如果出现根号下小于零的情况，直接赋值0，计算部分判断简单些（nan判定很烦……）
            if((idx>=1*nx)&&(idx<2*nx)){
				if(Ekall(Result[idx-nx])>=0.0)
					Result[idx] = Px(double(Result[idx-nx]));
				else Result[idx] = 0.0;
			}
			//第三列为第一列各自的fx初值，出现小于零情况同理。
			if((idx>=2*nx)&&(idx<3*nx)){
				if(Result[idx-1*nx]>0.0)
					Result[idx] = fx(double(Result[idx-2*nx]));
				else Result[idx] = 0.0;
			}
			
			//第四五六列为前三列的复制，为了compute函数准备
			if((idx>=3*nx)&&(idx<4*nx)){
				if(Result[idx-2*nx]>0.0)
					Result[idx] = Result[idx-3*nx];
				else Result[idx] = 0.0;
			}
			if((idx>=4*nx)&&(idx<5*nx))
				Result[idx] = Result[idx-3*nx];
			
		}
	}

}



void NormalRandom(double *ip, const int size){
    

	hiprandGenerator_t gen;                                  //生成随机数变量
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);//步骤1：指定算法
    hiprandSetPseudoRandomGeneratorSeed(gen, 11ULL);         //步骤2：随机数初始化
    hiprandGenerateNormalDouble(gen, ip, size, 0, 0.7);        //步骤3：生成随机数，存储到缓冲器中（第1个数字为均值，第二个为方差）
    hiprandDestroyGenerator(gen);                         	//释放指针
	return;
	
	
}


void  InitialMatrix(double* d_Result,int nx,int ny){
	NormalRandom(d_Result,nx);
	//分配grid,block大小
	int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	InitialKernel<<<grid,block>>>(d_Result,nx,ny);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	
	
	
	//保存数据仅仅为了测试用，写好compute部分以后肯定不用保存这个数据了……
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	double *h_gpuRef;
	h_gpuRef = (double *)malloc(nBytes);
	CHECK(hipMemcpy(h_gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));
	//保存数据
	double iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"init.dat");
	double iElaps = seconds() - iStart;
    printf("STORE THE InitialKernel DATA elapsed %lf sec\n",iElaps);
	
	
}


__global__ void ComputeKernel(double* Result,int nx,int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int idxOfXi  = 3 * nx + ix;
	unsigned int idxOfPxi = 4 * nx + ix;
	unsigned int idxOfXiTwo  = 5 * nx + ix;
	unsigned int idxOfPxiTwo = 6 * nx + ix;
	unsigned int idxOfTemp = 7 * nx + ix;


    if(ix<nx && Result[idxOfXi]!=0.0){
		for(int i=0;i<STEPSFIRST;i++){
			updateXi(Result[idxOfXi],Result[idxOfPxi]);
		}
		Result[idxOfXiTwo] = Result[idxOfXi];
		Result[idxOfPxiTwo] = Result[idxOfPxi];
		
		for(int i=0;i<STEPSSECOND;i++){
			updateXiAtStepTwo(Result[idxOfXiTwo],Result[idxOfPxiTwo],i*DX);
		}
		
		double TempE=0.5 * (pow(Result[idxOfPxiTwo],2.0)) - (1.0 / sqrt( pow(Result[idxOfXiTwo],2.0)+ pow(A,2.0)));
		if( TempE <= 0.0)
			Result[idxOfTemp]=-999;
	}
}



int CountZeros(double* h_Result,int nx)
{

	unsigned int idxOfXi  = nx ;
	unsigned int idxOfTemp = 7 * nx ;
	int count=0;
	for(int i=0;i<nx;i++){
		if(h_Result[idxOfXi+i] == 0.0f) count++;
		//if(h_Result[idxOfTemp+i] == -999) nonZeros++;
	}
	
	
	
	return count;
}

int CountTooBig(double* h_Result,int nx)
{

	unsigned int idxOfXi  = nx ;
	unsigned int idxOfTemp = 7 * nx ;
	int count=0;
	for(int i=0;i<nx;i++){
		//if(h_Result[idxOfXi+i] == 0.0) count++;
		if(h_Result[idxOfTemp+i] == -999) count++;
	}
	
	
	
	return count;
}









 void ComputeOnGPU1(double* Result,int nx,int ny,double* h_gpuRef){
	
	
	//分配grid,block大小
	int dimx = 512;
    dim3 block(dimx);
    dim3 grid((nx + block.x - 1) / block.x, 1);
	
	double iStart = seconds();
	ComputeKernel<<<grid,block>>>(Result,nx,ny);
	 CHECK(hipDeviceSynchronize());
	//如果核函数错误，返回信息
    CHECK(hipGetLastError());
	double iElaps = seconds() - iStart;
	printf("RungeOnGPU  elapsed %f sec\n",iElaps);
	
	// GPU数据拷贝回主机
	int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
	CHECK(hipMemcpy(h_gpuRef, Result, nBytes, hipMemcpyDeviceToHost));
	
	int zeros=0,nonzeros=0;
	zeros = CountZeros(h_gpuRef,nx);
	nonzeros = CountTooBig(h_gpuRef,nx);
	printf("The Number of Zeros is %d,\t The Number of NonZeros is %d \n",zeros,nonzeros);
	double per = (nx - zeros - nonzeros)/(nx - zeros);
	printf("Percentage is %lf  \n",per);
	
	//保存数据
	iStart = seconds();
	StoreData(h_gpuRef,nx,ny,"gpuStepTwo1202.dat");
	//StoreData(h_Random,1,ny,"h_Random.dat");
	iElaps = seconds() - iStart;
    printf("STORE THE ComputeKernel DATA elapsed %lf sec\n",iElaps);
	return;
}

