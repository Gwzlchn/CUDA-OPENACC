#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define PI    3.14159265358979323846 
#define rd (rand()/(RAND_MAX+1.0))  //此处是关于随机生成正态分布的定义

__device__ const double A=1;
__device__ const double E0=0.5;

//步长DX 终点TIME
__device__ const double DX=0.0027;
__device__ const int TOSTOP=10000;

//#include<cutil_math.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. A 1D thread block and 1D grid are used. sumArraysOnHost sequentially
 * iterates through vector elements on the host.
 */

 void StoreData(double *Matrix, const int NX,const int NY,const char name[])
{
   
	
	FILE* fp;
	fp = fopen(name, "w");
    if (!fp)
    {
        perror("cannot open file");
	}
	
    for (int i = 0; i < NX; i++)
    {
		
		for(int j=0;j<NY;j++){
			fprintf(fp,"%-.10lf\t\t",*(Matrix+j*NX+i));
			
		}
		fprintf(fp,"\n");
       
    }

	return;
}

//区间[min,max]上的均匀分布
double rand_m(double min, double max)
{
    return min+(max-min)*rand()/(RAND_MAX+1.0);
}

//求均值为miu，方差为sigma的正态分布函数在x处的函数值
double normal(double x, double miu,double sigma)
{
    return 1.0/sqrt(2*PI)/sigma*exp(-1*(x-miu)*(x-miu)/(2*sigma*sigma));
}

//按照矩形区域在函数值曲线上下位置分布情况得到正态分布函数x值
double rand_normal_distribution(double miu,double sigma, double min ,double max)
{
    double x,y,dScope;
    do{
        x=rand_m(min,max);
        y=normal(x,miu,sigma);
        dScope=rand_m(0.0,normal(miu,miu,sigma));
    }while(dScope>y);
    return x;
}

void initialData(double *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = rand_normal_distribution(0.0,1.0,-10.0,10.0);
    }
	
    return;
}

__device__ double f(double x,double y)
{
	return cos(x);
}

/*__device__ double rk4(double dx, double x, double y)
{
	double	k1 = dx * f(x, y),
		k2 = dx * f(x + dx / 2, y + k1 / 2),
		k3 = dx * f(x + dx / 2, y + k2 / 2),
		k4 = dx * f(x + dx, y + k3);
	return y + (k1 + 2 * k2 + 2 * k3 + k4) / 6;
}*/



__device__ double fx(double x)
{

	return -1.0/(pow(sqrt(pow(x,2.0)+pow(A,2.0)),3.0));
}

__device__ double Ekall(double x)
{

	return E0+1.0/(sqrt(pow(x,2.0)+pow(A,2.0)));
}

__device__ double Px(double x)
{
	return sqrt(2*Ekall(x));
}

__device__ double updateXi(double xi,double dx)
{
	double tempPx=Px(xi);
	double K1=tempPx,
		K2=xi+(tempPx/2.0)*dx,
		K3=xi+(((tempPx/2.0)*dx)+xi)/2.0*dx,
		K4=xi+(((((tempPx/2.0)*dx)+xi)/2.0*dx)+xi)*dx;
	
	return xi+dx*(K1+2*K2+2*K3+K4)/6.0;
}

__device__ double updatePxi(double xi,double dx)
{
	double tempPx=Px(xi),tempFx=fx(xi);
	double K1=tempFx,
		K2=tempPx+(tempFx/2.0)*dx,
		K3=tempPx+(((tempFx/2.0)*dx)+tempPx)/2.0*dx,
		K4=tempPx+(((((tempFx/2.0)*dx)+xi)/2.0*dx)+tempPx)*dx;
	
	return tempPx+dx*(K1+2*K2+2*K3+K4)/6.0;
}


// grid 1D block 1D
__global__ void RungeOnGPU1D(double *MatA,double *Result, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx ){
		
            //int idx = iy * nx + ix;
             //Result[idx] = MatA[idx];
			 //Result[1*nx] = MatA[nx];
        
        for (int iy = 0; iy <ny; iy++)
        {
            int idx = iy * nx + ix;
            if((idx<=1*nx))
				Result[idx] = MatA[idx];
			if((idx>=1*nx)&&(idx<2*nx))
				Result[idx] = Px(double(MatA[idx-nx]));
			if((idx>=2*nx)&&(idx<3*nx))
				Result[idx] = fx(double(MatA[idx-2*nx]));
			if((idx>=3*nx)&&(idx<4*nx)){
				Result[idx] = Result[idx-3*nx];
				int i,n=1+(TOSTOP)/DX;
				for(i=1;i<n;i++)
					Result[idx]=updateXi(Result[idx],DX);
			}
			if((idx>=4*nx)&&(idx<5*nx)){
				Result[idx] = Result[idx-3*nx];
				int i,n=1+(TOSTOP)/DX;
				for(i=1;i<n;i++)
					Result[idx]=updatePxi(Result[idx],DX);
			}
			
				
				/*const double dx=0.00001;
				int i,n=1+(2*PI)/dx;
				double temp;
				for (i = 1; i < n; i++){
					temp=rk4(dx, Result[idx-2*nx] + dx * (i - 1), Result[idx]);
					Result[idx] = temp;*/
					
		}
	}

}
		
	
	

	



int main()
{
    FILE* TIME_USED;
	TIME_USED = fopen("TimeData", "w");
    if (!TIME_USED)
    {
        perror("cannot open file");
	}
	
	
	printf("Starting...\n");
	fprintf(TIME_USED,"Starting...\n");

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
	fprintf(TIME_USED,"Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 10;
    int ny = 1 << 3;
	
	
	
	

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(double);
    printf("Matrix size: nx %d ny %d\n", nx, ny);
	fprintf(TIME_USED,"Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    double *h_Random, *gpuRef;
   
    
    gpuRef = (double *)malloc(nBytes);
    h_Random = (double *)malloc(nBytes);

    // initialize data at host side
    double iStart = seconds();
    initialData(h_Random, nxy);
    double iElaps = seconds() - iStart;
    printf("initialize matrix elapsed %f sec\n", iElaps);
	fprintf(TIME_USED,"initialize matrix elapsed %f sec\n", iElaps);

   
    memset(gpuRef, 0, nBytes);

   
	
	

    // malloc device global memory
    double *d_Random, *d_Result;
    CHECK(hipMalloc((void **)&d_Random, nBytes));
    
    CHECK(hipMalloc((void **)&d_Result, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_Random, h_Random, nBytes, hipMemcpyHostToDevice));
    

    // invoke kernel at host side
    int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);

    iStart = seconds();
    RungeOnGPU1D<<<grid, block>>>(d_Random, d_Result, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", 
			grid.x,grid.y,block.x, block.y, iElaps);
	fprintf(TIME_USED,"sumMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", 
			grid.x,grid.y,block.x, block.y, iElaps);
    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_Result, nBytes, hipMemcpyDeviceToHost));

    // check device results
    //checkResult(hostRef, gpuRef, nxy);

	
	
	
	
	

	//Store DATA	 
	iStart = seconds();
	StoreData(gpuRef,nx,ny,"gpu.dat");
	StoreData(h_Random,nx,1,"h_Random.dat");
	iElaps = seconds() - iStart;
    printf("STORE THE DATA elapsed %lf sec\n",iElaps);
	fprintf(TIME_USED,"STORE THE DATA elapsed %lf sec\n",iElaps);
    
	
	
	// free device global memory
    CHECK(hipFree(d_Random));
    

    // free host memory
    free(h_Random);
   
    
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
